#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *out, const int *in) {
            thrust::device_vector<int> dev_in(in, in + n);
            thrust::device_vector<int> dev_out(n);

            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:

            thrust::exclusive_scan(dev_in.begin(), dev_in.end(), dev_out.begin());
            timer().endGpuTimer();
            
            thrust::copy(dev_out.begin(), dev_out.end(), out);
        }
    }
}
