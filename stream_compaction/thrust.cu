#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            thrust::device_vector<int> thrust_dv_idata(idata, idata + n);
            thrust::device_vector<int> thrust_dv_odata(odata, odata + n);

            timer().startGpuTimer();

            thrust::exclusive_scan(thrust_dv_idata.begin(), thrust_dv_idata.end(), thrust_dv_odata.begin());

            timer().endGpuTimer();

            hipMemcpy(odata, (thrust_dv_odata.data()).get(), sizeof(int) * n, hipMemcpyDeviceToHost);
        }
    }
}
