#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_out;
            int* dev_in;
            hipMalloc((void**)&dev_out, n * sizeof(int));
            hipMalloc((void**)&dev_in, n * sizeof(int));
            hipMemcpy(dev_out, odata, sizeof(int) * n, hipMemcpyHostToDevice);
            hipMemcpy(dev_in, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            thrust::device_ptr<int> dev_thrust_out(dev_out);
            thrust::device_ptr<int> dev_thrust_in(dev_in);
            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
            thrust::exclusive_scan(dev_thrust_in, dev_thrust_in + n, dev_thrust_out);
            timer().endGpuTimer();
            hipMemcpy(odata, dev_out, sizeof(int) * n, hipMemcpyDeviceToHost);
            hipFree(dev_out);
            hipFree(dev_in);
        }
    }
}
