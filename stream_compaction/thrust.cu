#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include "common.h"
#include "thrust.h"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
            int* dev_idata;
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc dev_idata failed!");
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorWithLine("hipMemcpy dev_idata failed!");

            int* dev_odata;
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc dev_odata failed!");

            thrust::device_ptr<int> dev_thrust_in = thrust::device_ptr<int>(dev_idata);
            thrust::device_ptr<int> dev_thrust_out = thrust::device_ptr<int>(dev_odata);

            // only time thrust scan
            timer().startGpuTimer();
            thrust::exclusive_scan(dev_thrust_in, dev_thrust_in + n, dev_thrust_out);
            timer().endGpuTimer();


            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorWithLine("cudaMempcy dev_odata failed!");
        }

        void sort(int n, int* odata, int* idata) {
            timer().startGpuTimer();

            int* dev_idata;
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc dev_idata failed!");
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorWithLine("hipMemcpy dev_idata failed!");

            thrust::device_ptr<int> dev_thrust_in = thrust::device_ptr<int>(dev_idata);

            thrust::sort(dev_thrust_in, dev_thrust_in + n);

            hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);

            timer().endGpuTimer();
        }
    }
}
