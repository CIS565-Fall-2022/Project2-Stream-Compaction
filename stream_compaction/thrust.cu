#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
            
            thrust::host_vector<int> host_thrust_idata(idata, idata+n);
            thrust::device_vector<int> dev_thrust_idata(host_thrust_idata);
            thrust::exclusive_scan(dev_thrust_idata.begin(), dev_thrust_idata.end(), dev_thrust_idata.begin());
            host_thrust_idata = dev_thrust_idata;
            timer().endGpuTimer();

            thrust::copy(host_thrust_idata.begin(), host_thrust_idata.end(), odata);
        }
    }
}
