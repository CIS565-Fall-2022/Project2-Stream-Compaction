#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata, bool enableTimer) {
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

            //int data[6] = { 1, 0, 2, 2, 1, 3 };
            //thrust::exclusive_scan(thrust::host, data, data + 6, data, 4); // in-place scan
            //// data is now {4, 5, 5, 7, 9, 10}

            thrust::device_vector<int> dInpVec(idata, idata + n);
            thrust::device_vector<int> dOutVec(n);
            thrust::host_vector<int> hOutVec(n);

            if (enableTimer) timer().startGpuTimer();

            thrust::exclusive_scan(dInpVec.begin(), dInpVec.end(), dOutVec.begin(), 0);

            if (enableTimer) timer().endGpuTimer();

            thrust::copy(dOutVec.begin(), dOutVec.end(), hOutVec.begin());
            std::memcpy(odata, hOutVec.data(), n * sizeof(int));
        }


        void sort(int n, int* odata, const int* idata, bool enableTimer) {
            thrust::device_vector<int> dInpVec(idata, idata + n);
            thrust::host_vector<int> hOutVec(n);

            if (enableTimer) timer().startGpuTimer();

            thrust::sort(dInpVec.begin(), dInpVec.end());

            if (enableTimer) timer().endGpuTimer();

            thrust::copy(dInpVec.begin(), dInpVec.end(), hOutVec.begin());
            std::memcpy(odata, hOutVec.data(), n * sizeof(int));
        }
    }
}
