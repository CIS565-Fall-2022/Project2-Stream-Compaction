#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 256

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int* data, int depth) {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= n) {
                return;
            }
            int offset = pow(2, depth + 1);
            if (index % offset == 0) {
                data[index + offset - 1] += data[index + (int)pow(2, depth) - 1];
            }
        }

        __global__ void kernDownSweep(int n, int* data, int depth) {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= n) {
                return;
            }
            int offset1 = pow(2, depth);
            int offset2 = pow(2, depth + 1);
            if (index % offset2 == 0) {
                int t = data[index + offset1 - 1];
                data[index + offset1 - 1] = data[index + offset2 - 1];
                data[index + offset2 - 1] += t;
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {       
            int power = ilog2ceil(n);
            int arraySize = pow(2, power);
            dim3 blockPerGrid((arraySize + blockSize - 1) / blockSize);
            dim3 threadPerBlock(blockSize);

            int* dev_data;

            // create memory
            hipMalloc((void**)&dev_data, arraySize * sizeof(int));
            // set data and then copy the original data 
            hipMemset(dev_data, 0, arraySize * sizeof(int));
            hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            // TODO
            for (int i = 0; i < power; ++i) {
                kernUpSweep << <blockPerGrid, threadPerBlock >> > (arraySize, dev_data, i);
            }
            // set the root to 0
            hipMemset(dev_data + arraySize - 1, 0, sizeof(int));
            for (int i = power - 1; i >= 0; --i) {
                kernDownSweep << <blockPerGrid, threadPerBlock >> > (arraySize, dev_data, i);
            }
            timer().endGpuTimer();
            hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);

            // free memory
            hipFree(dev_data);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int power = ilog2ceil(n);
            int arraySize = pow(2, power);
            dim3 blockPerGrid((arraySize + blockSize - 1) / blockSize);
            dim3 threadPerBlock(blockSize);

            int* dev_idata;
            int* dev_odata;
            int* dev_boolBuffer;
            int* dev_scanResultBuffer;

            // malloc
            hipMalloc((void**)&dev_idata, arraySize * sizeof(int));
            hipMalloc((void**)&dev_odata, arraySize * sizeof(int));
            hipMalloc((void**)&dev_boolBuffer, arraySize * sizeof(int));
            hipMalloc((void**)&dev_scanResultBuffer, arraySize * sizeof(int));

            // set data and copy data
            // important for non power of two data!
            // if not set to 0, when map to boolean, the extra data which is not 0 will cause damage
            hipMemset(dev_idata , 0, arraySize * sizeof(int)); 
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            // TODO
            StreamCompaction::Common::kernMapToBoolean << <blockPerGrid, threadPerBlock>> > (arraySize, 
                                                                                                                                                                              dev_boolBuffer,
                                                                                                                                                                              dev_idata);
            hipMemcpy(dev_scanResultBuffer, dev_boolBuffer, arraySize * sizeof(int), hipMemcpyDeviceToDevice);

            for (int i = 0; i < power; ++i) {
                kernUpSweep << <blockPerGrid, threadPerBlock >> > (arraySize, dev_scanResultBuffer, i);
            }
            // set the root to 0
            hipMemset(dev_scanResultBuffer + arraySize - 1, 0, sizeof(int));
            for (int i = power - 1; i >= 0; --i) {
                kernDownSweep << <blockPerGrid, threadPerBlock >> > (arraySize, dev_scanResultBuffer, i);
            }

            StreamCompaction::Common::kernScatter << <blockPerGrid, threadPerBlock >> > (arraySize,
                                                                                                                                                                 dev_odata, dev_idata,
                                                                                                                                                                 dev_boolBuffer, dev_scanResultBuffer);
            timer().endGpuTimer();

            int* host_scanResultBuffer = new int[arraySize];
            hipMemcpy(host_scanResultBuffer, dev_scanResultBuffer, arraySize * sizeof(int), hipMemcpyDeviceToHost);

            int resultCount = host_scanResultBuffer[arraySize - 1];
            hipMemcpy(odata, dev_odata, resultCount * sizeof(int), hipMemcpyDeviceToHost);
            return resultCount;
        }
    }
}
