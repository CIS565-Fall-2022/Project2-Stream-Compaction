#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include "iostream"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        /*! Block size used for CUDA kernel launch. */
        #define blockSize 128

        // fix this and use new function1
        __global__ void reductionHelper(int n, int offset, int* tdata) {
            int idx = threadIdx.x + (blockIdx.x * blockDim.x);
            if (idx < n && (idx + 1) % offset == 0) {
                int neighborLoc = offset / 2;
                int a = tdata[idx - neighborLoc];
                int b = tdata[idx];
                tdata[idx] = a + b;

                //printf("red: %i, \n", a + b);
            }
        }

        // use function from class
        __global__ void partialSumHelper(int n, int offset, int* tdata) {
            int idx = threadIdx.x + (blockIdx.x * blockDim.x);
            if (idx < n && (idx + 1) % offset == 0) {
                int neighborLoc = offset / 2;
                int a = tdata[idx - neighborLoc];
                int b = tdata[idx];

                tdata[idx - neighborLoc] = b;
                tdata[idx] = a + b;
            }
        }

        // shift all nums one to the left
        __global__ void shiftLeft(int n, int* odata, int* idata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index < n) {
                if (index > 0) {
                    odata[index - 1] = idata[index];
                }
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

            int fullBlocksPerArray = (n + blockSize - 1) / blockSize;


            // shift tidata to the right to prepend 0's.
            int nextPowTwo = ilog2ceil(n);
            int numZeroes = pow(2, nextPowTwo) - n;

            // 1. up sweep same as reduction
            // empty buffer as idata
            // malloc enough space for n and 0's
            int* dev_tidata;
            hipMalloc((void**)&dev_tidata, (n + numZeroes) * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc dev_tifailed failed!");

            // set all elems to 0
            hipMemset(dev_tidata, 0, (n + numZeroes) * sizeof(int));
            checkCUDAErrorWithLine("hipMemset all elems to 0 failed!");

            // copy contents of idata into tidata so we can just pass in tidata and modify that on every pass.
            // antyhing after n + numZeroes is all 0's
            hipMemcpy(dev_tidata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorWithLine("hipMemcpy idata to dev_tidata failed!");

            int depth = 0;

            for (depth = 1; depth <= ilog2ceil(n + numZeroes); depth++) {
                int offset = pow(2, depth);
                reductionHelper << <fullBlocksPerArray, blockSize>>>(n + numZeroes, offset, dev_tidata);
                // wait for cuda timer. wait for all threads to finish
                hipDeviceSynchronize();
            }

            // store the last int of the array temporarily
            int lastInt = 0;
            hipMemcpy(&lastInt, dev_tidata + n + numZeroes - 1, sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorWithLine("hipMemcpy lastInt Device to Host failed!");

            // set last int of array to 0
            hipMemset(&dev_tidata[n + numZeroes - 1], 0, sizeof(int));
            checkCUDAErrorWithLine("hipMemset last int failed!");

            // 2. down sweep
            // takes dev_tidata as input
            for (depth; depth >= 1; depth--) {
                int offset = pow(2, depth);
                partialSumHelper << <fullBlocksPerArray, blockSize >> > (n + numZeroes, offset, dev_tidata);
                hipDeviceSynchronize();
            }

            // create output array for shifting
            int* dev_tidataFinal;
            hipMalloc((void**)&dev_tidataFinal, (n + numZeroes) * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc dev_tidataFinal failed!");

            // shift entire list to the left by 1 to remove extraneous 0 at the beginning
            shiftLeft << <fullBlocksPerArray, blockSize >> > (n + numZeroes, dev_tidataFinal, dev_tidata);

            // set last number in the array
            hipMemcpy(dev_tidata + n + numZeroes - 1, &lastInt, sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorWithLine("hipMemcpy lastInt Host to Device failed!");

            // copy final result to odata
            hipMemcpy(odata, dev_tidata, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorWithLine("hipMemcpy dev_tidata to odata failed!");

            // free all buffers
            hipFree(dev_tidata);
            hipFree(dev_tidataFinal);

            timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

            // 1. compute temp array
            
            // 2. exclusive scan
            // 3. scatter
            
            timer().endGpuTimer();
            return -1;
        }
    }
}
