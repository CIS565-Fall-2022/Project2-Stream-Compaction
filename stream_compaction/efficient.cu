#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        #define blockSize 8

        int* dev_idata;
        int* dev_odata;
        int* dev_buf;

        __global__ void upSweep(int N, int* idata, int* odata, int depth) {
            int k = threadIdx.x + (blockIdx.x * blockDim.x);
            if (k >= N) {
                return;
            }
            
            if ((k+1)%(1 << depth) == 0) {
                odata[k] = idata[k] + idata[k - (1 << (depth-1))];
            }
            else {
                odata[k] = idata[k];
            }

        }

        void zeroArray(int n, int* a) {
            for (int i = 0; i < n; i++) {
                a[i] = 0;
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            int arrLen;
            int maxDepth = ilog2ceil(n);
            maxDepth > ilog2(n) ? arrLen = pow(2, maxDepth) : arrLen = n;
            zeroArray(arrLen, odata);

            dim3 threadsPerBlock(arrLen / blockSize);

            int* buf = new int[arrLen];

            for (int i = 0; i < arrLen; i++) {
                if (i < n) {
                    buf[i] = idata[i];
                }
                else {
                    buf[i] = 0;
                }
            }

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            hipMalloc((void**)&dev_odata, arrLen * sizeof(int));
            hipMalloc((void**)&dev_buf, arrLen * sizeof(int));

            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            hipMemcpy(dev_odata, odata, sizeof(int) * arrLen, hipMemcpyHostToDevice);
            hipMemcpy(dev_buf, buf, sizeof(int) * arrLen, hipMemcpyHostToDevice);

            for (int i = 1; i <= maxDepth; i++) {
                upSweep << <threadsPerBlock, blockSize >> > (arrLen, dev_buf, dev_odata, i);
                hipMemcpy(dev_buf, dev_odata, sizeof(int) * arrLen, hipMemcpyDeviceToDevice);
            }

            hipMemcpy((void**)idata, dev_idata, sizeof(int) * n, hipMemcpyDeviceToHost);
            hipMemcpy(odata, dev_odata, sizeof(int) * arrLen, hipMemcpyDeviceToHost);
            hipMemcpy(buf, dev_buf, sizeof(int) * arrLen, hipMemcpyDeviceToHost);

            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_buf);


            timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
