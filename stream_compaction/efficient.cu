#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define SCAN_EFFI_REDUCE_BANK_CONFLICT 1

namespace StreamCompaction {
    namespace Efficient {
        enum class ScanSource { Host, Device };

        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __device__ inline int bankOffset(int idx) {
            return idx >> 5;
        }
        __device__ inline int offsetAddr(int idx) {
#if SCAN_EFFI_REDUCE_BANK_CONFLICT
            return idx + bankOffset(idx);
#else
            return idx;
#endif
        }

        __global__ void kernPartialUpSweep(int* data, int n, int stride) {
            int idx = blockDim.x * blockIdx.x + threadIdx.x + 1;
            if (idx > n) {
                return;
            }
            int mappedIdx = idx * stride - 1;
            data[mappedIdx] += data[mappedIdx - stride / 2];
        }

        __global__ void kernPartialDownSweep(int* data, int n, int stride) {
            int idx = blockDim.x * blockIdx.x + threadIdx.x + 1;
            if (idx > n) {
                return;
            }
            int mappedIdx = idx * stride - 1;
            data[mappedIdx] += data[mappedIdx - stride / 2];
            data[mappedIdx - stride / 2] = data[mappedIdx] - data[mappedIdx - stride / 2];
        }

        __global__ void kernBlockScanShared(int* data, int* blockSum, int n) {
            extern __shared__ int shared[];
            extern __shared__ int last;

            int idx = threadIdx.x + 1;
            int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;

            if (globalIdx > n) {
                return;
            }

            shared[offsetAddr(idx - 1)] = data[globalIdx];
            if (idx == blockDim.x) {
                last = shared[offsetAddr(blockDim.x - 1)];
            }
            __syncthreads();
#pragma unroll
            for (int stride = 1, active = blockDim.x >> 1; stride < blockDim.x; stride <<= 1, active >>= 1) {
                if (idx <= active) {
                    int idxPa = offsetAddr(idx * stride * 2 - 1);
                    int idxCh = offsetAddr(idx * stride * 2 - 1 - stride);
                    shared[idxPa] += shared[idxCh];
                }
                __syncthreads();
            }

            if (idx == 1) {
                shared[offsetAddr(blockDim.x - 1)] = 0;
            }
            __syncthreads();
#pragma unroll
            for (int stride = blockDim.x >> 1, active = 1; stride >= 1; stride >>= 1, active <<= 1) {
                if (idx <= active) {
                    int idxPa = offsetAddr(idx * stride * 2 - 1);
                    int idxCh = offsetAddr(idx * stride * 2 - 1 - stride);
                    shared[idxPa] += shared[idxCh];
                    shared[idxCh] = shared[idxPa] - shared[idxCh];
                }
                __syncthreads();
            }
            data[globalIdx] = shared[offsetAddr(idx - 1)];

            if (idx == 1) {
                blockSum[blockIdx.x] = shared[offsetAddr(blockDim.x - 1)] + last;
            }
        }

        __global__ void kernScannedBlockAdd(int* data, const int* blockSum, int n) {
            extern __shared__ int sum;
            int idx = blockDim.x * blockIdx.x + threadIdx.x;
            if (idx >= n) {
                return;
            }

            if (threadIdx.x == 0) {
                sum = blockSum[blockIdx.x];
            }
            __syncthreads();
            data[idx] += sum;
        }

        void devScanInPlace(int* devData, int size) {
            if (size != ceilPow2(size)) {
                throw std::runtime_error("devScanInPlace:: size not pow of 2");
            }

            for (int stride = 2; stride <= size; stride <<= 1) {
                int num = size / stride;
                int blockSize = Common::getDynamicBlockSizeEXT(num);
                int blockNum = ceilDiv(num, blockSize);
                kernPartialUpSweep<<<blockNum, blockSize>>>(devData, num, stride);
            }

            hipMemset(devData + size - 1, 0, sizeof(int));
            for (int stride = size; stride >= 2; stride >>= 1) {
                int num = size / stride;
                int blockSize = Common::getDynamicBlockSizeEXT(num);
                int blockNum = ceilDiv(num, blockSize);
                kernPartialDownSweep<<<blockNum, blockSize>>>(devData, num, stride);
            }
        }

        void devBlockScanInPlaceShared(int* devData, int* devBlockSum, int size, int blockSize) {
            if (size % blockSize != 0) {
                throw std::runtime_error("devBlockScanInPlaceShared:: size not multiple of BlockSize");
            }
            kernBlockScanShared<<<size / blockSize, blockSize>>>(devData, devBlockSum, size);
        }

        void devScanInPlaceShared(int* devData, int size) {
            if (size % SharedScanBlockSize != 0 || size <= SharedScanBlockSize) {
                throw std::runtime_error("devScanInPlaceShared:: size not multiple of BlockSize");
            }

            DevSharedScanAuxBuffer<int> devBuf(size, SharedScanBlockSize);
            hipMemcpy(devBuf.data(), devData, size * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToDevice);

            timer().startGpuTimer();
            for (int i = 0; i + 1 < devBuf.numLayers(); i++) {
                devBlockScanInPlaceShared(devBuf[i], devBuf[i + 1], devBuf.sizeAt(i), SharedScanBlockSize);
            }

            for (int i = devBuf.numLayers() - 2; i > 0; i--) {
                devScannedBlockAdd(devBuf[i - 1], devBuf[i], devBuf.sizeAt(i - 1), SharedScanBlockSize);
            }
            timer().endGpuTimer();

            hipMemcpy(devData, devBuf.data(), size * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToDevice);
            devBuf.destroy();
        }

        void devScannedBlockAdd(int* devData, int* devBlockSum, int n, int blockSize) {
            if (n % blockSize != 0) {
                throw std::runtime_error("devScannedBlockAdd:: size not multiple of BlockSize");
            }
            kernScannedBlockAdd<<<n / blockSize, blockSize>>>(devData, devBlockSum, n);
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // TODO
            int size = ceilPow2(n);
            int* data;
            hipMalloc(&data, size * sizeof(int));
            hipMemcpy(data, idata, n * sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice);

            timer().startGpuTimer();

            devScanInPlace(data, size);

            timer().endGpuTimer();
            
            hipMemcpy(odata, data, n * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
            hipFree(data);
        }

        void scanShared(int* out, const int* in, int n, int blockSize) {
            // Just to keep the edge case correct
            // If n <= blockSize, there's no need to perform a GPU scan
            if (n <= blockSize) {
                out[0] = 0;
                for (int i = 1; i < n; i++) {
                    out[i] = out[i - 1] + in[i - 1];
                }
                return;
            }

            DevSharedScanAuxBuffer<int> devBuf(n, SharedScanBlockSize);
            hipMemcpy(devBuf.data(), in, n * sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice);

            timer().startGpuTimer();
            for (int i = 0; i + 1 < devBuf.numLayers(); i++) {
                devBlockScanInPlaceShared(devBuf[i], devBuf[i + 1], devBuf.sizeAt(i), blockSize);
            }

            for (int i = devBuf.numLayers() - 2; i > 0; i--) {
                devScannedBlockAdd(devBuf[i - 1], devBuf[i], devBuf.sizeAt(i - 1), blockSize);
            }
            timer().endGpuTimer();

            hipMemcpy(out, devBuf.data(), n * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
            devBuf.destroy();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param out    The array into which to store elements.
         * @param in     The array of elements to compact.
         * @param n      The number of elements in idata.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int* out, const int* in, int n)
        {
            int* devIn, * devOut;
            hipMalloc(&devIn, n * sizeof(int));
            hipMalloc(&devOut, n * sizeof(int));
            hipMemcpy(devIn, in, n * sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice);

            int size = ceilPow2(n);
            int* devIndices;
            hipMalloc(&devIndices, size * sizeof(int));

            timer().startGpuTimer();

            int blockSize = Common::getDynamicBlockSizeEXT(n);
            int blockNum = ceilDiv(n, blockSize);

            Common::kernMapToBoolean<<<blockNum, blockSize>>>(n, devIndices, devIn);
            devScanInPlace(devIndices, size);
            Common::kernScatter<<<blockNum, blockSize>>>(n, devOut, devIn, devIn, devIndices);

            timer().endGpuTimer();

            int compactedSize;
            hipMemcpy(&compactedSize, devIndices + n - 1, sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
            compactedSize += (in[n - 1] != 0);

            hipMemcpy(out, devOut, compactedSize * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);

            hipFree(devIndices);
            hipFree(devIn);
            hipFree(devOut);

            return compactedSize;
        }

        int compactShared(int* out, const int* in, int n)
        {
            int* devIn, * devOut;
            hipMalloc(&devIn, n * sizeof(int));
            hipMalloc(&devOut, n * sizeof(int));
            hipMemcpy(devIn, in, n * sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice);

            int size = ceilPow2(n);
            int* devIndices;
            hipMalloc(&devIndices, size * sizeof(int));

            DevSharedScanAuxBuffer<int> devBuf(n, SharedScanBlockSize);

            timer().startGpuTimer();

            int blockSize = Common::getDynamicBlockSizeEXT(n);
            int blockNum = ceilDiv(n, blockSize);
            Common::kernMapToBoolean<<<blockNum, blockSize>>>(n, devIndices, devIn);
            hipMemcpy(devBuf.data(), devIndices, n * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToDevice);

            for (int i = 0; i + 1 < devBuf.numLayers(); i++) {
                devBlockScanInPlaceShared(devBuf[i], devBuf[i + 1], devBuf.sizeAt(i), SharedScanBlockSize);
            }
            for (int i = devBuf.numLayers() - 2; i > 0; i--) {
                devScannedBlockAdd(devBuf[i - 1], devBuf[i], devBuf.sizeAt(i - 1), SharedScanBlockSize);
            }

            hipMemcpy(devIndices, devBuf.data(), n * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToDevice);
            Common::kernScatter<<<blockNum, blockSize>>>(n, devOut, devIn, devIn, devIndices);

            timer().endGpuTimer();

            int compactedSize;
            hipMemcpy(&compactedSize, devIndices + n - 1, sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
            compactedSize += (in[n - 1] != 0);

            hipMemcpy(out, devOut, compactedSize * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
            hipFree(devIndices);
            hipFree(devIn);
            hipFree(devOut);
            devBuf.destroy();

            return compactedSize;
        }
    }
}
