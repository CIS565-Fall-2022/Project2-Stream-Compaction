#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        enum class ScanSource { Host, Device };

        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __device__ inline int bankOffset(int idx, int stride) {
            return ((idx & 0b11111) * stride) >> 5;
        }

        __global__ void kernPartialUpSweep(int* data, int n, int stride) {
            int idx = blockDim.x * blockIdx.x + threadIdx.x + 1;
            if (idx > n) {
                return;
            }
            int mappedIdx = idx * stride - 1;
            data[mappedIdx] += data[mappedIdx - stride / 2];
        }

        __global__ void kernPartialDownSweep(int* data, int n, int stride) {
            int idx = blockDim.x * blockIdx.x + threadIdx.x + 1;
            if (idx > n) {
                return;
            }
            int mappedIdx = idx * stride - 1;
            data[mappedIdx] += data[mappedIdx - stride / 2];
            data[mappedIdx - stride / 2] = data[mappedIdx] - data[mappedIdx - stride / 2];
        }

        __global__ void kernBlockScanShared(int* data, int* blockSum, int n) {
            extern __shared__ int shared[];

            int idx = threadIdx.x + 1;
            int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;

            if (globalIdx > n) {
                return;
            }

            shared[idx - 1] = data[globalIdx];
            if (idx == blockDim.x) {
                shared[idx] = shared[idx - 1];
            }
            __syncthreads();
#pragma unroll
            for (int stride = 1, active = blockDim.x >> 1; stride < blockDim.x; stride <<= 1, active >>= 1) {
                if (idx <= active) {
                    int mappedIdx = idx * stride * 2 - 1;
                    shared[mappedIdx] += shared[mappedIdx - stride];
                }
                __syncthreads();
            }

            if (idx == 1) {
                shared[blockDim.x - 1] = 0;
            }
            __syncthreads();
#pragma unroll
            for (int stride = blockDim.x >> 1, active = 1; stride >= 1; stride >>= 1, active <<= 1) {
                if (idx <= active) {
                    int mappedIdx = idx * stride * 2 - 1;
                    shared[mappedIdx] += shared[mappedIdx - stride];
                    shared[mappedIdx - stride] = shared[mappedIdx] - shared[mappedIdx - stride];
                }
                __syncthreads();
            }
            data[globalIdx] = shared[idx - 1];

            if (idx == 1) {
                blockSum[blockIdx.x] = shared[blockDim.x - 1] + shared[blockDim.x];
            }
        }

        __global__ void kernScannedBlockAdd(int* data, const int* blockSum, int n) {
            extern __shared__ int sum;
            int idx = blockDim.x * blockIdx.x + threadIdx.x;
            if (idx >= n) {
                return;
            }

            if (threadIdx.x == 0) {
                sum = blockSum[blockIdx.x];
            }
            __syncthreads();
            data[idx] += sum;
        }

        void devScanInPlace(int* devData, int size) {
            if (size != ceilPow2(size)) {
                throw std::runtime_error("devScanInPlace:: size not pow of 2");
            }

            for (int stride = 2; stride <= size; stride <<= 1) {
                int num = size / stride;
                int blockSize = Common::getDynamicBlockSizeEXT(num);
                int blockNum = ceilDiv(num, blockSize);
                kernPartialUpSweep<<<blockNum, blockSize>>>(devData, num, stride);
            }

            hipMemset(devData + size - 1, 0, sizeof(int));
            for (int stride = size; stride >= 2; stride >>= 1) {
                int num = size / stride;
                int blockSize = Common::getDynamicBlockSizeEXT(num);
                int blockNum = ceilDiv(num, blockSize);
                kernPartialDownSweep<<<blockNum, blockSize>>>(devData, num, stride);
            }
        }

        void devBlockScanInPlaceShared(int* devData, int* devBlockSum, int size, int blockSize) {
            if (size % blockSize != 0) {
                throw std::runtime_error("devBlockScanInPlaceShared:: size not multiple of BlockSize");
            }
            kernBlockScanShared<<<size / blockSize, blockSize>>>(devData, devBlockSum, size);
        }

        void devScanInPlaceShared(int* devData, int size) {
            const int blockSize = 128;
            if (size % blockSize != 0 || size <= blockSize) {
                throw std::runtime_error("devScanInPlaceShared:: size not multiple of BlockSize");
            }

            std::vector<DevMemRec<int>> sums;
            for (int i = size; i >= 1; i = ceilDiv(i, blockSize)) {
                int size = ceilDiv(i, blockSize) * blockSize;
                int* sum;
                hipMalloc(&sum, size * sizeof(int));
                sums.push_back({ sum, size, i });

                if (i == 1) {
                    break;
                }
            }
            hipMemcpy(sums[0].ptr, devData, size * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToDevice);

            for (int i = 0; i + 1 < sums.size(); i++) {
                devBlockScanInPlaceShared(sums[i].ptr, sums[i + 1].ptr, sums[i].size, blockSize);
            }

            for (int i = sums.size() - 2; i > 0; i--) {
                kernScannedBlockAdd<<<sums[i].size, blockSize>>>(sums[i - 1].ptr, sums[i].ptr, sums[i - 1].size);
            }
            hipMemcpy(devData, sums[0].ptr, size * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToDevice);

            for (auto& sum : sums) {
                hipFree(sum.ptr);
            }
        }

        void devScannedBlockAdd(int* devData, int* devBlockSum, int n, int blockSize) {
            if (n % blockSize != 0) {
                throw std::runtime_error("devScannedBlockAdd:: size not multiple of BlockSize");
            }
            kernScannedBlockAdd<<<n / blockSize, blockSize>>>(devData, devBlockSum, n);
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // TODO
            int size = ceilPow2(n);
            int* data;
            hipMalloc(&data, size * sizeof(int));
            hipMemcpy(data, idata, n * sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice);

            timer().startGpuTimer();

            devScanInPlace(data, size);

            timer().endGpuTimer();
            
            hipMemcpy(odata, data, n * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
            hipFree(data);
        }

        void scanShared(int* out, const int* in, int n, int blockSize) {
            // Just to keep the edge case correct
            // If n <= blockSize, there's no need to perform a GPU scan
            if (n <= blockSize) {
                out[0] = 0;
                for (int i = 1; i < n; i++) {
                    out[i] = out[i - 1] + in[i - 1];
                }
                return;
            }

            std::vector<DevMemRec<int>> sums;
            for (int i = n; i >= 1; i = ceilDiv(i, blockSize)) {
                int size = ceilDiv(i, blockSize) * blockSize;
                int* sum;
                hipMalloc(&sum, size * sizeof(int));
                sums.push_back({ sum, size, i });

                if (i == 1) {
                    break;
                }
            }
            hipMemcpy(sums[0].ptr, in, n * sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice);

            timer().startGpuTimer();
            for (int i = 0; i + 1 < sums.size(); i++) {
                devBlockScanInPlaceShared(sums[i].ptr, sums[i + 1].ptr, sums[i].size, blockSize);
            }

            for (int i = sums.size() - 2; i > 0; i--) {
                devScannedBlockAdd(sums[i - 1].ptr, sums[i].ptr, sums[i - 1].size, blockSize);
            }
            timer().endGpuTimer();

            hipDeviceSynchronize();
            hipMemcpy(out, sums[0].ptr, n * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);

            for (auto& sum : sums) {
                hipFree(sum.ptr);
            }
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param out    The array into which to store elements.
         * @param in     The array of elements to compact.
         * @param n      The number of elements in idata.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int* out, const int* in, int n)
        {
            int* devIn, * devOut;
            hipMalloc(&devIn, n * sizeof(int));
            hipMalloc(&devOut, n * sizeof(int));
            hipMemcpy(devIn, in, n * sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice);

            int size = ceilPow2(n);
            int* devIndices;
            hipMalloc(&devIndices, size * sizeof(int));

            timer().startGpuTimer();

            int blockSize = Common::getDynamicBlockSizeEXT(n);
            int blockNum = ceilDiv(n, blockSize);

            Common::kernMapToBoolean<<<blockNum, blockSize>>>(n, devIndices, devIn);
            devScanInPlace(devIndices, size);
            Common::kernScatter<<<blockNum, blockSize>>>(n, devOut, devIn, devIn, devIndices);

            timer().endGpuTimer();

            int compactedSize;
            hipMemcpy(&compactedSize, devIndices + n - 1, sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
            compactedSize += (in[n - 1] != 0);

            hipMemcpy(out, devOut, compactedSize * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);

            hipFree(devIndices);
            hipFree(devIn);
            hipFree(devOut);

            return compactedSize;
        }

        int compactShared(int* out, const int* in, int n)
        {
            const int ScanBlockSize = 128;
            int* devIn, * devOut;
            hipMalloc(&devIn, n * sizeof(int));
            hipMalloc(&devOut, n * sizeof(int));
            hipMemcpy(devIn, in, n * sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice);

            int size = ceilPow2(n);
            int* devIndices;
            hipMalloc(&devIndices, size * sizeof(int));

            std::vector<DevMemRec<int>> sums;
            for (int i = n; i >= 1; i = ceilDiv(i, ScanBlockSize)) {
                int sz = ceilDiv(i, ScanBlockSize) * ScanBlockSize;
                int* sum;
                hipMalloc(&sum, sz * sizeof(int));
                sums.push_back({ sum, sz, i });

                if (i == 1) {
                    break;
                }
            }
            timer().startGpuTimer();

            int blockSize = Common::getDynamicBlockSizeEXT(n);
            int blockNum = ceilDiv(n, blockSize);

            Common::kernMapToBoolean<<<blockNum, blockSize>>>(n, devIndices, devIn);
            hipMemcpy(sums[0].ptr, devIndices, n * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToDevice);

            for (int i = 0; i + 1 < sums.size(); i++) {
                devBlockScanInPlaceShared(sums[i].ptr, sums[i + 1].ptr, sums[i].size, ScanBlockSize);
            }

            for (int i = sums.size() - 2; i > 0; i--) {
                devScannedBlockAdd(sums[i - 1].ptr, sums[i].ptr, sums[i - 1].size, ScanBlockSize);
            }

            hipMemcpy(devIndices, sums[0].ptr, n * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToDevice);
            Common::kernScatter<<<blockNum, blockSize>>>(n, devOut, devIn, devIn, devIndices);

            timer().endGpuTimer();

            int compactedSize;
            hipMemcpy(&compactedSize, devIndices + n - 1, sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
            compactedSize += (in[n - 1] != 0);

            hipMemcpy(out, devOut, compactedSize * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
            hipFree(devIndices);
            hipFree(devIn);
            hipFree(devOut);

            for (auto& sum : sums) {
                hipFree(sum.ptr);
            }
            return compactedSize;
        }
    }
}
