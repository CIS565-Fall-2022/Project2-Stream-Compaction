#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 128

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int d, int* x)
        {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }
            if (index % (1 << (d+1)) == 0) 
            {

                x[index + (1 << (d + 1)) - 1] += x[index + (1 << d ) - 1];
            }
        }

        __global__ void kernDownSweep(int n, int d, int* x)
        {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            if (index % (1 << (d + 1)) == 0)
            {

                int t = x[index + (1 << d) - 1];
                x[index + (1 << d) - 1] = x[index + (1 << (d + 1)) - 1];
                x[index + (1 << (d + 1)) - 1] += t;
            }
        }
        void upDownSweep(int n, int* data, dim3 blockPerGrid)
        {
            for (int d = 0; d <= ilog2ceil(n) - 1; ++d) {
                kernUpSweep << < blockPerGrid, blockSize >> > (n, d, data);
            }
            hipDeviceSynchronize();

            hipMemset(data + n - 1, 0, sizeof(int));
            checkCUDAError("hipMemset failed!");


            for (int d = ilog2ceil(n) - 1; d >= 0; --d) {
                kernDownSweep << < blockPerGrid, blockSize >> > (n, d, data);
            }
            hipDeviceSynchronize();
        }


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int* odata, const int* idata) {
            int intermArraySize = 1 << ilog2ceil(n);
            dim3 fullBlocksPerGrid((blockSize + intermArraySize - 1) / blockSize);

            int* dev_data;
            hipMalloc((void**)&dev_data, intermArraySize * sizeof(int));
            checkCUDAError("hipMalloc dev_data failed!");
            hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);

            timer().startGpuTimer();

            upDownSweep(intermArraySize, dev_data, fullBlocksPerGrid);
            
            timer().endGpuTimer();
            hipMemcpy(odata, dev_data, sizeof(int) * n, hipMemcpyDeviceToHost);
            hipFree(dev_data);

        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int* odata, const int* idata) {
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
            int arraySize = 1 << ilog2ceil(n);

            int* dev_indices;
            int* dev_bool;
            int* dev_idata;
            int* dev_odata;
            hipMalloc((void**)&dev_indices, arraySize * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed!");
            hipMalloc((void**)&dev_bool, arraySize * sizeof(int));
            checkCUDAError("hipMalloc dev_bool failed!");
            hipMalloc((void**)&dev_idata, arraySize * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy dev_bool to dev_data failed!");


            timer().startGpuTimer();
            // Step 1
            StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (n, dev_bool, dev_idata);
            hipDeviceSynchronize();
            
            hipMemcpy(dev_indices, dev_bool, n * sizeof(int), hipMemcpyDeviceToDevice);
            checkCUDAError("hipMemcpy dev_bool to dev_data failed!");


            // Step 2
            upDownSweep(arraySize, dev_indices, fullBlocksPerGrid);

            int returnSize = 0;
            hipMemcpy(&returnSize, dev_indices + arraySize - 1, sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_indices to host failed!");

            // Di shared this edge case with me
            // When the input array has the last element non-zero, it will fail
            // hence we can add the last bit of the bool array to the return size to make sure that this case is covered
            int lastBool = 0;
            hipMemcpy(&lastBool, dev_bool + arraySize - 1, sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_bool to host failed!");
            returnSize += lastBool;


            hipMalloc((void**)&dev_odata, returnSize * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");


            // Step 3
             StreamCompaction::Common::kernScatter <<<fullBlocksPerGrid, blockSize >>>(arraySize, dev_odata,
                 dev_idata, dev_bool, dev_indices);
             hipDeviceSynchronize();


            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, returnSize * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_odata to host failed!");
            
            hipFree(dev_indices);
            hipFree(dev_odata);
            hipFree(dev_idata);
            hipFree(dev_bool);

            return returnSize;
        }
    }
}
