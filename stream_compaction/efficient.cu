#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 256

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        __global__ void kernUpSweep(int n, int* data, int offset)
        {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            if (index % (2 * offset) == 0) {
                int desIdx = index + (2 * offset) - 1;
                int srcIdx = index + offset - 1;

                data[desIdx] += data[srcIdx];
            }
        }

        __global__ void kernDownSweep(int n, int* data, int offset)
        {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }
            if (index % (2 * offset) == 0) {
                int t = data[index + offset - 1];
                data[index + offset - 1] = data[index + offset * 2 - 1];
                data[index + offset * 2 - 1] += t;
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) 
        {
            int maxDepth = ilog2ceil(n);
            int maxSize = pow(2, maxDepth);
            dim3 fullBlocksPerGrid((maxSize + blockSize - 1) / blockSize);
            
            int *dev_data;
            hipMalloc((void**)&dev_data, maxSize * sizeof(int));
            hipMemcpy(dev_data, idata, maxSize * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            // TODO
            // UpSweep
            for (int d = 0; d < maxDepth; d++) {
                kernUpSweep << < fullBlocksPerGrid, blockSize >> > (maxSize, dev_data, pow(2, d));
            }

            hipMemset(dev_data + maxSize - 1, 0, sizeof(int));

            // DownSweep
            for (int d = maxDepth - 1; d >= 0; d--) {
                kernDownSweep << < fullBlocksPerGrid, blockSize >> > (maxSize, dev_data, pow(2, d));
            }

            timer().endGpuTimer();

            hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);

            // free cuda memory
            hipFree(dev_data);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) 
        {
            int *dev_idata, *dev_odata, *dev_bool, *dev_idx;

            int maxDepth = ilog2ceil(n);
            int maxSize = pow(2, maxDepth);

            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
            dim3 maxBlocksPerGrid((maxSize + blockSize - 1) / blockSize);

            hipMalloc((void**)&dev_idata, maxSize * sizeof(int));
            hipMalloc((void**)&dev_odata, maxSize * sizeof(int));
            hipMalloc((void**)&dev_bool, maxSize * sizeof(int));
            hipMalloc((void**)&dev_idx, maxSize * sizeof(int));

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            // TODO

            Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (n, dev_bool, dev_idata);
            hipMemcpy(dev_idx, dev_bool, maxSize * sizeof(int), hipMemcpyDeviceToDevice);

            // Scan
            // UpSweep
            for (int d = 0; d <= maxDepth - 1; d++) {
                kernUpSweep << < maxBlocksPerGrid, blockSize >> > (maxSize, dev_idx, pow(2, d));
            }

            hipMemset(dev_idx + maxSize - 1, 0, sizeof(int));

            // DownSweep
            for (int d = maxDepth - 1; d >= 0; d--) {
                kernDownSweep << < maxBlocksPerGrid, blockSize >> > (maxSize, dev_idx, pow(2, d));
            }

            // Scatter
            //scatter
            Common::kernScatter << < fullBlocksPerGrid, blockSize >> > (maxSize, dev_odata, dev_idata, dev_bool, dev_idx);


            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);

            // compute num of non-zero element
            int* arr = new int[maxSize];
            hipMemcpy(arr, dev_bool, sizeof(int) * maxSize, hipMemcpyDeviceToHost);

            int count = 0;
            for (int i = 0; i < maxSize; i++) {
                if (arr[i] == 1) {
                    count++;
                }
            }
            // Free cuda memory
            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_bool);
            hipFree(dev_idx);
            
            return count;
        }
    }
}
