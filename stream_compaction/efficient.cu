#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 128

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int d, int* x)
        {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }
            if (index % (1 << (d+1)) == 0) 
            {

                x[index + (1 << (d + 1)) - 1] += x[index + (1 << d ) - 1];
            }
        }

        __global__ void kernDownSweep(int n, int d, int* x)
        {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            if (index % (1 << (d + 1)) == 0)
            {

                int t = x[index + (1 << d) - 1];
                x[index + (1 << d) - 1] = x[index + (1 << (d + 1)) - 1];
                x[index + (1 << (d + 1)) - 1] += t;
            }
        }


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int* odata, const int* idata) {


            int intermArraySize = 1 << ilog2ceil(n);
            dim3 fullBlocksPerGrid((blockSize + intermArraySize - 1) / blockSize);


            int* dev_data;
            hipMalloc((void**)&dev_data, intermArraySize * sizeof(int));
            checkCUDAError("hipMalloc dev_data failed!");
            hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);

            timer().startGpuTimer();
            for (int d = 0; d <= ilog2ceil(intermArraySize) - 1; ++d) {
                kernUpSweep << < fullBlocksPerGrid, blockSize >> > (intermArraySize, d, dev_data);
                //hipDeviceSynchronize();
            }

            hipMemset(dev_data + intermArraySize - 1, 0, sizeof(int));
            checkCUDAError("hipMemset failed!");


            for (int d = ilog2ceil(intermArraySize) - 1; d >= 0; --d) {
                kernDownSweep << < fullBlocksPerGrid, blockSize >> > (intermArraySize, d, dev_data);
                //hipDeviceSynchronize();
            }


            timer().endGpuTimer();
            hipMemcpy(odata, dev_data, sizeof(int) * n, hipMemcpyDeviceToHost);
            hipFree(dev_data);

        }

        void scanRecursion(int n, int* data, dim3 blockPerGrid)
        {
            for (int d = 0; d <= ilog2ceil(n) - 1; ++d) {
                kernUpSweep << < blockPerGrid, blockSize >> > (n, d, data);
            }

            hipMemset(data + n - 1, 0, sizeof(int));
            checkCUDAError("hipMemset failed!");


            for (int d = ilog2ceil(n) - 1; d >= 0; --d) {
                kernDownSweep << < blockPerGrid, blockSize >> > (n, d, data);
            }

        }




        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int* odata, const int* idata) {
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            int* dev_data;
            int* dev_bool;
            hipMalloc((void**)&dev_data, n * sizeof(int));
            checkCUDAError("hipMalloc dev_data failed!");
            hipMalloc((void**)&dev_bool, n * sizeof(int));
            checkCUDAError("hipMalloc dev_data failed!");

            timer().startGpuTimer();
            // Step 1
            StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (n, dev_bool, idata);
            hipDeviceSynchronize();
            
            hipMemcpy(dev_data, dev_bool, sizeof(int) * n, hipMemcpyDeviceToDevice);

            // Step 2
            scanRecursion(1 << ilog2ceil(n), dev_data, fullBlocksPerGrid);

            // Step 3
             StreamCompaction::Common::kernScatter <<<fullBlocksPerGrid, blockSize >>>(n, odata,
                idata, dev_bool, dev_data);


            timer().endGpuTimer();
            int returnSize = 0;
            hipMemcpy(&returnSize, dev_data + n - 1, sizeof(int), hipMemcpyDeviceToHost);


            hipFree(dev_data);
            hipFree(dev_bool);

            return returnSize;
        }
    }
}
