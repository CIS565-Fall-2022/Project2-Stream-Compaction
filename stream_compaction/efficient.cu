#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpStreamReduction(int n, int *odata, const int *idata) {
            
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            int* dev_buffer1;
            int* dev_buffer2;
            int* dev_backup;

            dim3 gridSize(32, 32);
            dim3 blockSize(32, 32);

            // Memory allocation
            hipMalloc((void**)&dev_buffer1, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_buffer2, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_odata failed!");
            hipMalloc((void**)&dev_backup, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_odata failed!");

            hipMemcpy(dev_buffer1, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            hipMemcpy(dev_backup, dev_buffer1, sizeof(int) * n, hipMemcpyDeviceToDevice);

            kernUpStreamReduction << <gridSize, blockSize >> > (n, dev_buffer2, dev_buffer1);

            hipFree(dev_buffer1);
            hipFree(dev_buffer2);
            hipFree(dev_backup);
            timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
