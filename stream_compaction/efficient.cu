#include "hip/hip_runtime.h"
#include <iostream>
#include <memory>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer() {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        __global__ void kernUpSweep(int n, int log2n, int* inp) {
            int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (idx >= n) return;
            int num = n / 2;
            for (int d = 0; d < log2n; d++) {
                __syncthreads();
                if (idx >= num) continue;
                int offset = 1 << d; // 1 << d is pow(2, d)
                int idxWrite = offset * 2 * (idx + 1) - 1;
                inp[idxWrite] = inp[idxWrite] + inp[idxWrite - offset];
                num /= 2;
            }
            //__syncthreads();
            //if (idx == n - 1) inp[n - 1] = 0;
            //�� doesn't work when n is large ��
        }

        __global__ void kernDownSweep(int n, int log2n, int* inp) {
            int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (idx >= n) return;
            int offset = n / 2;
            for (int d = 0; d < log2n; d++) {
                __syncthreads();
                if (idx < 1 << d) {
                    int idxWrite = n - 1 - idx * offset * 2;
                    int tmp = inp[idxWrite];
                    inp[idxWrite] += inp[idxWrite - offset];
                    inp[idxWrite - offset] = tmp;
                }
                offset /= 2;
            }
        }

        void scan(int n, int *odata, const int *idata, bool enableTimer) {
            int* devInp;
            hipMalloc((void**)&devInp, n * sizeof(int));
            checkCUDAError("hipMalloc devInp failed!");
            hipMemcpy(devInp, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata failed!");

            if (enableTimer) timer().startGpuTimer();
            dim3 fullBlocksPerGrid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);
            kernUpSweep<<<fullBlocksPerGrid, BLOCK_SIZE>>>(n, ilog2ceil(n), devInp);
            hipMemset(devInp+n-1, 0, sizeof(int));
            kernDownSweep<<<fullBlocksPerGrid, BLOCK_SIZE>>>(n, ilog2ceil(n), devInp);
            if (enableTimer) timer().endGpuTimer();

            hipMemcpy(odata, devInp, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy odata failed!");
            hipFree(devInp);
            checkCUDAError("hipFree devInp failed!");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */


        int compact(int n, int *odata, const int *idata, bool enableTimer) {
            //int* devInp;
            //int* devBools;
            //int* devOut;
            //hipMalloc((void**)&devInp, n * sizeof(int));
            //checkCUDAError("hipMalloc devInp failed!");
            //hipMalloc((void**)&devOut, n * sizeof(int));
            //checkCUDAError("hipMalloc devOut failed!");
            //hipMalloc((void**)&devBools, n * sizeof(int));
            //checkCUDAError("hipMalloc devBools failed!");
            //hipMemcpy(devInp, idata, n * sizeof(int), hipMemcpyHostToDevice);
            //checkCUDAError("hipMemcpy idata failed!");

            //if (enableTimer) timer().startGpuTimer();
            //dim3 fullBlocksPerGrid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);
            //Common::kernMapToBoolean<<<fullBlocksPerGrid, BLOCK_SIZE>>> (n, devBools, devInp);
            //kernUpSweep<<<fullBlocksPerGrid, BLOCK_SIZE>>>(n, ilog2ceil(n), devBools);
            //hipMemset(devBools +n-1, 0, sizeof(int));
            //kernDownSweep<<<fullBlocksPerGrid, BLOCK_SIZE>>>(n, ilog2ceil(n), devBools);
            //Common::kernScatter<<<fullBlocksPerGrid, BLOCK_SIZE>>>(n, devOut, devInp, devBools);
            //if (enableTimer) timer().endGpuTimer();

            //hipMemcpy(odata, devOut, n * sizeof(int), hipMemcpyDeviceToHost);
            //checkCUDAError("hipMemcpy odata failed!");
            //std::unique_ptr<int[]> indices{ new int[n] };
            //hipMemcpy(indices.get(), devBools, n * sizeof(int), hipMemcpyDeviceToHost);
            //checkCUDAError("hipMemcpy devBools failed!");
            //hipFree(devInp);
            //checkCUDAError("hipFree devInp failed!");
            //hipFree(devOut);
            //checkCUDAError("hipFree devInp failed!");
            //hipFree(devBools);
            //checkCUDAError("hipFree devBools failed!");
            ////for (int i = 0; i < 32; i++) {
            ////    std::cout << indices[i] << " ";
            ////}
            ////std::cout << std::endl;
            //return idata[n - 1] != 0 ? indices[n - 1] + 1 : indices[n - 1];
            return -1;
        }
    }
}
