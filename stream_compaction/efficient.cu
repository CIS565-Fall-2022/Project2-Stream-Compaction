#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweepOpt(int n, int depth, int offset, int* data)
        {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            int offset_1 = offset << 1;
            if (index > n)
            {
                return;
            }
            data[index * offset_1 + offset_1 - 1] += data[index * offset_1 + offset - 1];

        }
        __global__ void kernDownSweepOpt(int n, int depth, int offset, int* data, bool root)
        {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            int offset_1 = offset << 1;
            if (index > n)
            {
                return;
            }
            if (index == (n - 1) && root)
            {
                data[index] = 0;
            }
            int t = data[index * offset_1 + offset -1];
            data[index * offset_1 + offset - 1] = data[index*offset_1 + offset_1 -1];
            data[index * offset_1 + offset_1 -1 ] += t;
          

        }

        __global__ void kernUpSweep(int n, int depth, int offset, int* data)
        {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index > n)
            {
                return;
            }
            if (((index + 1) % (1 << (depth + 1))) == 0)
            {
                data[index] += data[index - offset];
            }

        }
        __global__ void kernDownSweep(int n, int depth, int offset, int* data, bool root)
        {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index > n)
            {
                return;
            }
            if (index == (n-1) && root)
            {
                data[index] = 0;
            }
            if (((index + 1) % (1 << (depth + 1))) == 0)
            {
                int t = data[index - offset];
                data[index - offset] = data[index];
                data[index] += t;
            }

        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scanNoOpt(int n, int *odata, const int *idata) {
            int blockSize = 128;
            int numBlocks = ((n + blockSize - 1) / blockSize);
            int* dev_data;
            float d = ilog2ceil(n);
            int pot = pow(2, d);
            hipMalloc((void**)&dev_data, pot * sizeof(int));
            hipMemset(dev_data, 0, pot * sizeof(int));
            hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            hipDeviceSynchronize();
            timer().startGpuTimer();
            for (int depth = 0; depth < d; depth++)
            {
                int offset = 1 << depth;
                kernUpSweep << < numBlocks, blockSize >> > (pot, depth, offset, dev_data);
                
            }
            bool root = true;
            hipDeviceSynchronize();
            for (int depth = d - 1; depth >= 0; depth--)
            {
                int offset = 1 << depth;
                kernDownSweep << < numBlocks, blockSize >> > (pot, depth, offset, dev_data, root);
                root = false;
            }
            timer().endGpuTimer();
            hipMemcpy(odata, dev_data, sizeof(int) * n, hipMemcpyDeviceToHost);
            hipFree(dev_data);
        }

        void scan(int n, int* odata, const int* idata) {
            int blockSize = 128;
            int numBlocks = ((n + blockSize - 1) / blockSize);
            int* dev_data;
            float d = ilog2ceil(n);
            int pot = pow(2, d);
            hipMalloc((void**)&dev_data, pot * sizeof(int));
            hipMemset(dev_data, 0, pot * sizeof(int));
            hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            hipDeviceSynchronize();
            int num = pot;
            timer().startGpuTimer();
            for (int depth = 0; depth < d; depth++)
            {
                int offset = 1 << depth;
                kernUpSweep << < numBlocks, blockSize >> > (num, depth, offset, dev_data);
                num /= 2;

            }
            bool root = true;
            hipDeviceSynchronize();
            for (int depth = d - 1; depth > 0; depth--)
            {
                int offset = 1 << depth;
                kernDownSweep << < numBlocks, blockSize >> > (num, depth, offset, dev_data, root);
                root = false;
                num *= 2;
            }
            timer().endGpuTimer();
            hipMemcpy(odata, dev_data, sizeof(int) * n, hipMemcpyDeviceToHost);
            hipFree(dev_data);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int blockSize = 128;
            int numBlocks = ((n + blockSize - 1) / blockSize);
            int* dev_idata;
            int* dev_odata;
            int* dev_bools;
            int* dev_indicies;
            float d = ilog2ceil(n);
            int pot = pow(2, d);
            hipMalloc((void**)&dev_idata, pot * sizeof(int));
            hipMalloc((void**)&dev_odata, pot * sizeof(int));
            hipMalloc((void**)&dev_bools, pot * sizeof(int));
            hipMalloc((void**)&dev_indicies, pot * sizeof(int));
            hipMemset(dev_idata, 0, pot * sizeof(int));
            hipMemset(dev_odata, 0, pot * sizeof(int));
            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            hipDeviceSynchronize();
            timer().startGpuTimer();

            Common::kernMapToBoolean << < numBlocks, blockSize >> > (n, dev_bools, dev_idata);
            hipMemcpy(dev_indicies, dev_bools, sizeof(int) * n, hipMemcpyDeviceToDevice);
            for (int depth = 0; depth < d; depth++)
            {
                int offset = 1 << depth;
                kernUpSweep << < numBlocks, blockSize >> > (pot, depth, offset, dev_indicies);

            }
            bool root = true;
            hipDeviceSynchronize();
            for (int depth = d - 1; depth >= 0; depth--)
            {
                int offset = 1 << depth;
                kernDownSweep << < numBlocks, blockSize >> > (pot, depth, offset, dev_indicies, root);
                root = false;
            }
            Common::kernScatter << < numBlocks, blockSize >> > (n, dev_odata, dev_idata, dev_bools, dev_indicies);
            timer().endGpuTimer();
            hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
            int num_elements = 0;
            for (int i = 0; i < n; i++)
            {
                if (odata[i] != 0)
                {
                    num_elements++;
                }
            }
            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_bools);
            hipFree(dev_indicies);
            return num_elements;
        }
    }
}
