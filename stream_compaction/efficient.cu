#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        __global__ void kernUpSweep(int n, int d, int* idata) {
            // Parallel Reduction
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            int k = index * (1 << (d + 1));
            idata[k + (1 << (d + 1)) - 1] += idata[k + (1 << d) - 1];
        }

        __global__ void kernDownSweep(int n, int d, int* idata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            int k = index * (1 << (d + 1));
            int t = idata[k + (1 << d) - 1];
            idata[k + (1 << d) - 1] = idata[k + (1 << (d + 1)) - 1];
            idata[k + (1 << (d + 1)) - 1] += t;
        }

        __global__ void kernZeroRoot(int n, int* idata) {
            // Root is last element
            idata[n - 1] = 0;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // Account for non-powers of 2 by padding by 0
            int paddedN = (1 << ilog2ceil(n));
            int* dev_idata;
            hipMalloc((void**)&dev_idata, paddedN * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            hipMemset(dev_idata + n, 0, (paddedN - n) * sizeof(int));
            hipDeviceSynchronize();

            timer().startGpuTimer();
            // Upsweep
            for (int i = 0; i < ilog2ceil(n); ++i) {
                int numThreads = paddedN / (1 << (i + 1));
                dim3 upSweepGridSize((numThreads + blockSize - 1) / blockSize);
                kernUpSweep << <upSweepGridSize, blockSize >> >
                    (numThreads, i, dev_idata);
                checkCUDAError("kernUpSweep failed!");
                hipDeviceSynchronize();
            }

            // Downsweep
            kernZeroRoot << <1, 1 >> > (paddedN, dev_idata);
            for (int i = ilog2ceil(n) - 1; i >= 0; --i) {
                int numThreads = paddedN / (1 << (i + 1));
                dim3 downSweepGridSize((numThreads + blockSize - 1) / blockSize);
                kernDownSweep << <downSweepGridSize, blockSize >> >
                    (numThreads, i, dev_idata);
                checkCUDAError("kernDownSweep failed!");
                hipDeviceSynchronize();
            }
            timer().endGpuTimer();
            hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(dev_idata);
        }



        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            // Account for non-powers of 2 by padding by 0
            int paddedN = (1 << ilog2ceil(n));
            int* dev_idata;
            int* dev_odata;
            int* dev_bool;
            int* dev_indices;

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            
            // Pad bool array instead of idata to save operations in kernMapToBoolean
            hipMalloc((void**)&dev_bool, paddedN * sizeof(int));
            checkCUDAError("hipMalloc dev_bool failed!");
            hipMemset(dev_bool + n, 0, (paddedN - n) * sizeof(int));

            hipMalloc((void**)&dev_indices, paddedN * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed!");
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed!");
            hipDeviceSynchronize();

            timer().startGpuTimer();
            // Binarize
            dim3 nGridSize((n + blockSize - 1) / blockSize);
            StreamCompaction::Common::kernMapToBoolean << < nGridSize, blockSize >> >
                (n, dev_bool, dev_idata);
            checkCUDAError("kernMapToBoolean failed!");
            hipDeviceSynchronize();
            // We need bool array for scatter so copy bool result to indices to be modified in place
            hipMemcpy(dev_indices, dev_bool, paddedN * sizeof(int), hipMemcpyDeviceToDevice);
            checkCUDAError("hipMemcpy failed!");
            hipDeviceSynchronize();
 
            // Copied Scan code from above
            // Upsweep
            for (int i = 0; i < ilog2ceil(n); ++i) {
                int numThreads = paddedN / (1 << (i + 1));
                dim3 upSweepGridSize((numThreads + blockSize - 1) / blockSize);
                kernUpSweep << <upSweepGridSize, blockSize >> >
                    (numThreads, i, dev_indices);
                checkCUDAError("kernUpSweep failed!");
                hipDeviceSynchronize();
            }

            // Downsweep
            kernZeroRoot << <1, 1 >> > (paddedN, dev_indices);
            for (int i = ilog2ceil(n) - 1; i >= 0; --i) {
                int numThreads = paddedN / (1 << (i + 1));
                dim3 downSweepGridSize((numThreads + blockSize - 1) / blockSize);
                kernDownSweep << <downSweepGridSize, blockSize >> >
                    (numThreads, i, dev_indices);
                checkCUDAError("kernDownSweep failed!");
                hipDeviceSynchronize();
            }

            // Scatter
            StreamCompaction::Common::kernScatter << <nGridSize, blockSize >> >
                (n, dev_odata, dev_idata, dev_bool, dev_indices);
            checkCUDAError("kernScatter failed!");
            hipDeviceSynchronize();
            timer().endGpuTimer();
            
            hipMemcpy(odata, dev_indices, paddedN * sizeof(int), hipMemcpyDeviceToHost);
            int finalNum = odata[paddedN - 1];
            hipMemcpy(odata, dev_odata, finalNum * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(dev_idata);
            hipFree(dev_bool);
            hipFree(dev_indices);
            hipFree(dev_odata);
            return finalNum;
        }
    }
}
