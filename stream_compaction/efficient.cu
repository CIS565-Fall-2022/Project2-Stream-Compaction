#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include "iostream"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        /*! Block size used for CUDA kernel launch. */
        #define blockSize 128

        // fix this and use new function1
        __global__ void kernReductionHelper(int n, int offset, int* tdata) {
            int idx = threadIdx.x + (blockIdx.x * blockDim.x);
            if (idx < n && (idx + 1) % offset == 0) {
                int neighborLoc = offset / 2;
                int a = tdata[idx - neighborLoc];
                int b = tdata[idx];
                tdata[idx] = a + b;
            }
        }

        // use function from class
        __global__ void kernPartialSumHelper(int n, int offset, int* tdata) {
            int idx = threadIdx.x + (blockIdx.x * blockDim.x);
            if (idx < n && (idx + 1) % offset == 0) {
                int neighborLoc = offset / 2;
                int a = tdata[idx - neighborLoc];
                int b = tdata[idx];

                tdata[idx - neighborLoc] = b;
                tdata[idx] = a + b;
            }
        }

        // shift all nums one to the left
        __global__ void kernShiftLeft(int n, int* odata, int* idata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index < n) {
                if (index > 0) {
                    odata[index - 1] = idata[index];
                }
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

            int fullBlocksPerArray = (n + blockSize - 1) / blockSize;

            // shift tidata to the right to prepend 0's.
            int nextPowTwo = ilog2ceil(n);
            int numZeroes = pow(2, nextPowTwo) - n;

            // 1. up sweep same as reduction
            // empty buffer as idata
            // malloc enough space for n and 0's
            int* dev_tidata;
            hipMalloc((void**)&dev_tidata, (n + numZeroes) * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc dev_tifailed failed!");

            // set all elems to 0
            hipMemset(dev_tidata, 0, (n + numZeroes) * sizeof(int));
            checkCUDAErrorWithLine("hipMemset all elems to 0 failed!");

            // copy contents of idata into tidata so we can just pass in tidata and modify that on every pass.
            // antyhing after n + numZeroes is all 0's
            hipMemcpy(dev_tidata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorWithLine("hipMemcpy idata to dev_tidata failed!");

            int depth = 0;

            for (depth = 1; depth <= ilog2ceil(n + numZeroes); depth++) {
                int offset = pow(2, depth);
                kernReductionHelper << <fullBlocksPerArray, blockSize>>>(n + numZeroes, offset, dev_tidata);
                // wait for cuda timer. wait for all threads to finish
                hipDeviceSynchronize();
            }

            // set last int of array to 0
            hipMemset(&dev_tidata[n + numZeroes - 1], 0, sizeof(int));
            checkCUDAErrorWithLine("hipMemset last int failed!");

            // 2. down sweep
            // takes dev_tidata as input
            for (depth; depth >= 1; depth--) {
                int offset = pow(2, depth);
                kernPartialSumHelper << <fullBlocksPerArray, blockSize >> > (n + numZeroes, offset, dev_tidata);
                hipDeviceSynchronize();
            }

            // create output array for shifting
            int* dev_tidataFinal;
            hipMalloc((void**)&dev_tidataFinal, (n + numZeroes) * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc dev_tidataFinal failed!");

            // copy final result to odata
            hipMemcpy(odata, dev_tidata, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorWithLine("hipMemcpy dev_tidata to odata failed!");

            // free all buffers
            hipFree(dev_tidata);
            hipFree(dev_tidataFinal);

            timer().endGpuTimer();
        }

        __global__ void kernMapToBoolean(int n, int* odata, int* idata) {
            int idx = threadIdx.x + (blockIdx.x * blockDim.x);
            if (idx < n) {
                if (idata[idx] != 0) {
                    // add 1 to out array
                    odata[idx] = 1;
                }
                else {
                    // add 0 to outarray
                    odata[idx] = 0;
                }
            }
        }

        __global__ void kernScatter(int n, int* odata, int* idata, int* tdata, int* sdata) {
            int idx = threadIdx.x + (blockIdx.x * blockDim.x);
            if (idx < n) {
                if (tdata[idx] == 1) {
                    int destinationIdx = sdata[idx];
                    odata[destinationIdx] = idata[idx];
                }
                // otherwise do not write
            }
        }

        // reimplement scan for compact
        void compactScan(int n, int* dev_odata, int* dev_idata) {
            int fullBlocksPerArray = (n + blockSize - 1) / blockSize;

            // shift tidata to the right to prepend 0's.
            int nextPowTwo = ilog2ceil(n);
            int numZeroes = pow(2, nextPowTwo) - n;

            // 1. up sweep same as reduction
            // empty buffer as idata
            // malloc enough space for n and 0's
            int* dev_tidata;
            hipMalloc((void**)&dev_tidata, (n + numZeroes) * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc dev_tifailed failed!");

            // set all elems to 0
            hipMemset(dev_tidata, 0, (n + numZeroes) * sizeof(int));
            checkCUDAErrorWithLine("hipMemset all elems to 0 failed!");

            // copy contents of idata into tidata so we can just pass in tidata and modify that on every pass.
            // antyhing after n + numZeroes is all 0's
            hipMemcpy(dev_tidata, dev_idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorWithLine("hipMemcpy idata to dev_tidata failed!");

            int depth = 0;

            for (depth = 1; depth <= ilog2ceil(n + numZeroes); depth++) {
                int offset = 1 << depth; // pow(2, depth);
                kernReductionHelper << <fullBlocksPerArray, blockSize >> > (n + numZeroes, offset, dev_tidata);
                // wait for cuda timer. wait for all threads to finish
                hipDeviceSynchronize();
            }

            // set last int of array to 0
            hipMemset(&dev_tidata[n + numZeroes - 1], 0, sizeof(int));
            checkCUDAErrorWithLine("hipMemset last int failed!");

            // 2. down sweep
            // takes dev_tidata as input
            for (depth; depth >= 1; depth--) {
                int offset = pow(2, depth);
                kernPartialSumHelper << <fullBlocksPerArray, blockSize >> > (n + numZeroes, offset, dev_tidata);
                hipDeviceSynchronize();
            }

            // copy final result to odata
            hipMemcpy(dev_odata, dev_tidata, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorWithLine("hipMemcpy dev_tidata to odata failed!");

            // free all buffers
            hipFree(dev_tidata);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

            int fullBlocksPerArray = (n + blockSize - 1) / blockSize;

            // 1. compute temp array: 1 for everything that fits rule. 0 otherwise.
            int* dev_iArray;
            int* dev_tempArray;

            hipMalloc((void**)&dev_tempArray, n * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc dev_tempArray failed!");

            hipMalloc((void**)&dev_iArray, n * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc dev_iArray failed!");

            hipMemcpy(dev_iArray, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorWithLine("hipMemcpy dev_iArray failed!");

            kernMapToBoolean << <fullBlocksPerArray, blockSize >> > (n, dev_tempArray, dev_iArray);

            // 2. exclusive scan on tempArray.
            int* dev_scanArray;

            hipMalloc((void**)&dev_scanArray, n * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc dev_scanArray failed!");

            compactScan(n, dev_scanArray, dev_tempArray);

            // 3. scatter
            // last element of numScatters is the length of scatterArray.
            int numScatters = 0;
            int validSlot = 0;
            hipMemcpy(&numScatters, dev_scanArray + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&validSlot, dev_tempArray + n - 1, sizeof(int), hipMemcpyDeviceToHost);

            numScatters += validSlot;
            checkCUDAErrorWithLine("hipMemcpy numScatters failed!");

            int* dev_scatterFinal;
            hipMalloc((void**)&dev_scatterFinal, numScatters * sizeof(int));
            checkCUDAErrorWithLine("hipMalloc dev_scatterFinal failed!");

            kernScatter << <fullBlocksPerArray, blockSize >> > (n, dev_scatterFinal, dev_iArray, dev_tempArray, dev_scanArray);

            // memcpy back from odata1 to odata
            hipMemcpy(odata, dev_scatterFinal, numScatters * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorWithLine("hipMemcpy dev_scatterFinal to odata failed!");

            timer().endGpuTimer();

            hipFree(dev_iArray);
            hipFree(dev_tempArray);
            hipFree(dev_scanArray);
            hipFree(dev_scatterFinal);

            return numScatters;
        }
    }
}
