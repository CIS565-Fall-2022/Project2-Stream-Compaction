#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 128

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // unlike naive impl, this one doesn't shift the array
        __global__ void kernPadArray(int n, int paddedLen, int* odata, const int* idata) {
          int index = threadIdx.x + (blockIdx.x * blockDim.x);
          if (index < n) {
            odata[index] = idata[index];
          }
          else if (index < paddedLen) {
            odata[index] = 0;
          }
        }

        __global__ void kernUpsweep(int numThreads, int readStride, int* data) {
          int index = threadIdx.x + (blockIdx.x * blockDim.x);
          if (index >= numThreads) {
            return;
          }

          int writeStride = readStride * 2;

          // Index of what element to write to is calculated using write stride
          int writeIndex = (writeStride * index) + writeStride - 1;
          int readIndex = (writeStride * index) + readStride - 1;

          data[writeIndex] += data[readIndex];
        }

        __global__ void kernDownsweep(int numThreads, int writeStride, int* data) {
          int index = threadIdx.x + (blockIdx.x * blockDim.x);
          if (index >= numThreads) {
            return;
          }

          int readStride = writeStride * 2;

          int leftChildIndex = index * readStride + writeStride - 1;
          int rightChildIndex = index * readStride + readStride - 1; // right child is also where parent is stored

          int temp = data[leftChildIndex];
          data[leftChildIndex] = data[rightChildIndex];
          data[rightChildIndex] += temp;
        }

        void scanImpl(int paddedLength, int* dev_idata) {
          // Build tree (upsweep)
          // readStride = 2^depth, where depth goes from 0... log2n - 1... the stride between elements we read and sum
          // writeStride = 2^(depth + 1)... the stride between indices of elements we store the sums in
          for (int readStride = 1; readStride < paddedLength; readStride *= 2) {
            int writeStride = readStride * 2;

            int numThreads = paddedLength / writeStride; // one thread per element we write to
            dim3 numBlocks((numThreads + blockSize - 1) / blockSize);

            kernUpsweep << <numBlocks, blockSize >> > (numThreads, readStride, dev_idata);
            hipDeviceSynchronize();
          }

          // Down sweep
          // In down sweep, children now read info from parent. So writeStride = readStride / 2
          // Write stride = n/2, n/4, ... 4, 2, 1, aka. 2^depth
          // Read stride = 2^(depth + 1)

          // First set parent to 0
          int zero = 0;
          hipMemcpy(dev_idata + paddedLength - 1, &zero, sizeof(int), hipMemcpyHostToDevice);

          for (int writeStride = paddedLength / 2; writeStride >= 1; writeStride = writeStride >> 1) {
            int readStride = writeStride * 2;

            // now launch 1 thread per element we read from
            int numThreads = paddedLength / readStride;
            dim3 numBlocks((numThreads + blockSize - 1) / blockSize);

            kernDownsweep << <numBlocks, blockSize >> > (numThreads, writeStride, dev_idata);
          }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            // Pad array
            int* dev_unpadded_idata;
            int* dev_idata;

            int exponent = ilog2ceil(n);
            int paddedLength = pow(2, exponent);
            dim3 fullBlocksPerGrid((paddedLength + blockSize - 1) / blockSize);

            hipMalloc((void**)&dev_unpadded_idata, n * sizeof(int));
            hipMalloc((void**)&dev_idata, paddedLength * sizeof(int));

            hipMemcpy(dev_unpadded_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorFn("Cuda memcpy idata no work");

            kernPadArray << <fullBlocksPerGrid, blockSize >> > (n, paddedLength, dev_idata, dev_unpadded_idata);

            scanImpl(paddedLength, dev_idata);
            hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_unpadded_idata);
            hipFree(dev_idata);

            timer().endGpuTimer();
        }

        __global__ void kernGetPaddedBoolArray(int n, int paddedLength, int* odata, const int* idata) {
          int index = threadIdx.x + (blockIdx.x * blockDim.x);
          if (index < n) {
            odata[index] = idata[index] == 0 ? 0 : 1;
          }
          else if (index < paddedLength) {
            odata[index] = 0;
          }
        }

        __global__ void kernScatter(int n, int* odata, int* idata, int *boolData, int *boolScan) {
          int index = threadIdx.x + (blockIdx.x * blockDim.x);
          if (index >= n) {
            return;
          }

          if (boolData[index] == 1) {
            int writeIndex = boolScan[index];
            odata[writeIndex] = idata[index];
          }
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            int* dev_boolData;
            int* dev_boolScan;
            int* dev_idata;
            int* dev_odata;
            int exponent = ilog2ceil(n);
            int paddedLength = pow(2, exponent);

            hipMalloc((void**)&dev_boolData, paddedLength * sizeof(int));
            hipMalloc((void**)&dev_boolScan, paddedLength * sizeof(int));
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            hipMalloc((void**)&dev_odata, n * sizeof(int)); // don't need padding for scatter step
            // copy input data
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            dim3 fullBlocksPerGrid((paddedLength + blockSize - 1) / blockSize);
            kernGetPaddedBoolArray << <fullBlocksPerGrid, blockSize >> > (n, paddedLength, dev_boolData, dev_idata);
            // copy dev_boolData since the scan implementation is destructive
            hipMemcpy(dev_boolScan, dev_boolData, paddedLength * sizeof(int), hipMemcpyDeviceToDevice);
            scanImpl(paddedLength, dev_boolScan);

            dim3 fullBlocksPerGridNonPadded((n + blockSize - 1) / blockSize);
            kernScatter << <fullBlocksPerGridNonPadded, blockSize >> >
              (n, dev_odata, dev_idata, dev_boolData, dev_boolScan);

            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);

            // How to get the total # of elements compacted:
            // Get last (index n - 1) element of dev_boolScan and last (index n - 1) element of dev_boolData
            // if dev_boolData[n - 1] = 0, then it's the last element
            // Otherwise it's the last element + 1
            
            int lastElementIsIncluded, lastBoolScanVal, resultLength;

            hipMemcpy(&lastElementIsIncluded, dev_boolData + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&lastBoolScanVal, dev_boolScan + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            
            resultLength = lastElementIsIncluded == 1 ? lastBoolScanVal + 1 : lastBoolScanVal;

            hipFree(dev_boolData);
            hipFree(dev_boolScan);
            hipFree(dev_idata);
            hipFree(dev_odata);

            timer().endGpuTimer();

            return resultLength;
        }
    }
}
