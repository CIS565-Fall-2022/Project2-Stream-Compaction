#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweepOpt(int n, int depth, int offset, int* data)
        {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            int offset_1 = offset << 1;
            int new_index = index * offset_1 + offset_1 - 1;
            if (index > (n-1))
            {
                return;
            }
            data[new_index] += data[index * offset_1 + offset - 1];

        }
        __global__ void kernDownSweepOpt(int n, int depth, int offset, int* data)
        {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            int offset_1 = offset << 1;
            int parent_i = index * offset_1 + offset_1 - 1;
            int left_i = index * offset_1 + offset - 1;
            if (index > (n-1))
            {
                return;
            }
            if (n == 1)
            {
                data[parent_i] = 0;
            }
            int t = data[left_i];
            data[left_i] = data[parent_i];
            data[parent_i] += t;
          

        }

        //__global__ void kernUpSweep(int n, int depth, int offset, int* data)
        //{
        //    int index = threadIdx.x + (blockIdx.x * blockDim.x);
        //    if (index > n)
        //    {
        //        return;
        //    }
        //    if (((index + 1) % (1 << (depth + 1))) == 0)
        //    {
        //        data[index] += data[index - offset];
        //    }

        //}
        //__global__ void kernDownSweep(int n, int depth, int offset, int* data, bool root)
        //{
        //    int index = threadIdx.x + (blockIdx.x * blockDim.x);
        //    if (index > n)
        //    {
        //        return;
        //    }
        //    if (index == (n-1) && root)
        //    {
        //        data[index] = 0;
        //    }
        //    if (((index + 1) % (1 << (depth + 1))) == 0)
        //    {
        //        int t = data[index - offset];
        //        data[index - offset] = data[index];
        //        data[index] += t;
        //    }

        //}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        //void scanNoOpt(int n, int *odata, const int *idata) {
        //    int blockSize = 128;
        //    int numBlocks = ((n + blockSize - 1) / blockSize);
        //    int* dev_data;
        //    float d = ilog2ceil(n);
        //    int pot = pow(2, d);
        //    hipMalloc((void**)&dev_data, pot * sizeof(int));
        //    hipMemset(dev_data, 0, pot * sizeof(int));
        //    hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);
        //    hipDeviceSynchronize();
        //    timer().startGpuTimer();
        //    for (int depth = 0; depth < d; depth++)
        //    {
        //        int offset = 1 << depth;
        //        kernUpSweep << < numBlocks, blockSize >> > (pot, depth, offset, dev_data);
        //        
        //    }
        //    bool root = true;
        //    hipDeviceSynchronize();
        //    for (int depth = d - 1; depth >= 0; depth--)
        //    {
        //        int offset = 1 << depth;
        //        kernDownSweep << < numBlocks, blockSize >> > (pot, depth, offset, dev_data, root);
        //        root = false;
        //    }
        //    timer().endGpuTimer();
        //    hipMemcpy(odata, dev_data, sizeof(int) * n, hipMemcpyDeviceToHost);
        //    hipFree(dev_data);
        //}

        void scan(int n, int* odata, const int* idata) {
            int blockSize = 64;
            int* dev_data;
            int d = ilog2ceil(n);
            int pot = 1 << d;
            int num = pot;
            hipMalloc((void**)&dev_data, pot * sizeof(int));
            checkCUDAError("Malloc dev_data Failed! ");
            hipMemset(dev_data, 0, pot * sizeof(int));
            checkCUDAError("Memset dev_data Failed! ");
            hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("Memcpy dev_data Failed! ");
            hipDeviceSynchronize();
            
            timer().startGpuTimer();
            
            for (int depth = 0; depth < d; depth++)
            {
                num /= 2;
                int offset = 1 << depth;
                int numBlocks = ((num + blockSize - 1) / blockSize);
                kernUpSweepOpt << < numBlocks, blockSize >> > (num, depth, offset, dev_data);

            }
            hipDeviceSynchronize();
            for (int depth = d - 1; depth >= 0; depth--)
            {
                int offset = 1 << depth;
                int numBlocks = ((num + blockSize - 1) / blockSize);
                kernDownSweepOpt << < numBlocks, blockSize >> > (num, depth, offset, dev_data);
                num *= 2;
            }
            timer().endGpuTimer();
            hipMemcpy(odata, dev_data, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("Memcpy dev_data back Failed! ");
            hipFree(dev_data);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int blockSize = 64;
            int numBlocks = ((n + blockSize - 1) / blockSize);
            int* dev_idata;
            int* dev_odata;
            int* dev_bools;
            int* dev_indicies;
            int d = ilog2ceil(n);
            int pot = 1 << d;
            int num = pot;
            hipMalloc((void**)&dev_idata, pot * sizeof(int));
            checkCUDAError("Malloc dev_idata Failed! ");
            hipMalloc((void**)&dev_odata, pot * sizeof(int));
            checkCUDAError("Malloc dev_odata Failed! ");
            hipMalloc((void**)&dev_bools, pot * sizeof(int));
            checkCUDAError("Malloc dev_bools Failed! ");
            hipMalloc((void**)&dev_indicies, pot * sizeof(int));
            checkCUDAError("Malloc dev_indices Failed! ");
            hipMemset(dev_idata, 0, pot * sizeof(int));
            checkCUDAError("Memset dev_idata Failed! ");
            hipMemset(dev_odata, 0, pot * sizeof(int));
            checkCUDAError("Memset dev_odata Failed! ");
            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("Memcpy dev_idata Failed! ");
            hipDeviceSynchronize();
            timer().startGpuTimer();

            Common::kernMapToBoolean << < numBlocks, blockSize >> > (n, dev_bools, dev_idata);
            hipMemcpy(dev_indicies, dev_bools, sizeof(int) * n, hipMemcpyDeviceToDevice);
            for (int depth = 0; depth < d; depth++)
            {
                num /= 2;
                int offset = 1 << depth;
                int numBlocks = ((num + blockSize - 1) / blockSize);
                kernUpSweepOpt << < numBlocks, blockSize >> > (num, depth, offset, dev_indicies);

            }
            hipDeviceSynchronize();
            for (int depth = d - 1; depth >= 0; depth--)
            {
                int offset = 1 << depth;
                int numBlocks = ((num + blockSize - 1) / blockSize);
                kernDownSweepOpt << < numBlocks, blockSize >> > (num, depth, offset, dev_indicies);
                num *= 2;
            }
            Common::kernScatter << < numBlocks, blockSize >> > (n, dev_odata, dev_idata, dev_bools, dev_indicies);
            timer().endGpuTimer();
            hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("Memcpy dev_odata Failed! ");
            int num_elements = 0;
            for (int i = 0; i < n; i++)
            {
                if (odata[i] != 0)
                {
                    num_elements++;
                }
            }
            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_bools);
            hipFree(dev_indicies);
            return num_elements;
        }
    }
}
