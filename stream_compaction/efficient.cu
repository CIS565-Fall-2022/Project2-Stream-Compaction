#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

const int blockSize = 128;

__device__ inline int twoPow(int d) {
    return (1 << (d));
}

inline int twoPowHost(int d) {
    return (1 << (d));
}

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int d, int *x) {
            int idx = blockDim.x * blockIdx.x + threadIdx.x;
            if (idx >= n) return;
            if (idx % twoPow(d + 1) == 0)
                x[idx + twoPow(d + 1) - 1] += x[idx + twoPow(d) - 1];
        }

        __global__ void kernDownSweep(int n, int d, int *x) {
            int idx = blockDim.x * blockIdx.x + threadIdx.x;
            if (idx >= n) return;
            if (idx % twoPow(d + 1) == 0) {
                int tmp = x[idx + twoPow(d) - 1];
                x[idx + twoPow(d) - 1] = x[idx + twoPow(d + 1) - 1];
                x[idx + twoPow(d + 1) - 1] += tmp;
            }
        }

        /**
         * �����ʵ���У�������������ĳ��Ȳ���2���ݴε�������Ὣ����չ����С��2���ݴδ�С��
         * �������ĺô��ǿ��Խ���������ֳɹ�ģ��ͬ�������飬���ڲ��м��㡣
         * �ڼ�����ǰ׺�ͺ󣬰Ѷ���Ĳ��֣�������3��Ԫ�أ���Ϊ0���ɡ�
         *
         * ������˵����UpSweep�׶Σ�ÿ���̴߳���һ������Ԫ�أ�
         * ������Ԫ�ص��±�����idx % 2^(d+1) == 0��
         * �����Ԫ�ص�ֵ������ǰ�������2^d��Ԫ�ص�Ԫ�ص�ֵ��
         * �����ͽ�ÿ������Ϊ2^d��Ԫ�ض�Ӧ�ĺͼ��������
         * �������һ��ִ��log2(size)�Σ�ÿһ�δ���ľ��붼����һ�ε�������
         *
         * ��DownSweep�׶Σ��Ƚ����һ��Ԫ����Ϊ0��Ȼ������һ�㿪ʼ��
         * ÿ���̴߳���һ������Ԫ�أ�������Ԫ�ص��±�����idx % 2^(d+1) == 0��
         * �����Ԫ�ص�ֵ����ǰ�������2^d��Ԫ�ص�Ԫ�ص�ֵ����������ǰ���ֵ�ӵ������ֵ�ϡ�
         * �����ͽ�ÿ������Ϊ2^d��Ԫ�ض�Ӧ�ĺʹ������ϴ��ݡ�
         * ͬ�����������Ҳ��ִ��log2(size)�Ρ�������������ǰ׺�;ͼ�������ˡ�
         */

        /**
         * In this implementation, if the length of the input array is not a power of 2,
         * it will be extended to the smallest power of 2 size.
         * This is done to facilitate parallel computation by dividing the input array into equally-sized subarrays.
         * After computing the prefix sum, the excess part of the array (i.e., the last 3 elements) is set to 0.
         *
         * Specifically, in the UpSweep phase, each thread processes one element of the array.
         * If the index of this element satisfies idx % 2^(d+1) == 0,
         * then the value of this element is added to the value of the element located 2^d positions in front of it.
         * This way, the sums of every two elements that are 2^d apart are calculated. This process is repeated log2(size) times,
         * where each iteration processes elements that are twice as far apart as the previous iteration.
         *
         * In the DownSweep phase, the last element of the array is set to 0.
         * Starting from the last level, each thread processes one element of the array.
         * If the index of this element satisfies idx % 2^(d+1) == 0,
         * then the value of this element is swapped with the value of the element located 2^d positions in front of it,
         * and the value of the latter element is added to the former element.
         * This way, the sums of every two elements that are 2^d apart are propagated upwards from the bottom of the array.
         * Again, this process is repeated log2(size) times.
         * Finally, the prefix sum of the entire array is computed.
         */

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int size = twoPowHost(ilog2ceil(n)); // ensure the size is pow of 2
            // for example:
            // if n = 253, let size equal to 256.
            // ilog2ceil(253) = [log2(253)] + 1 = log2(128) + 1 = 8
            // twoPowHost(8) = 256
            dim3 blockPerGrids((size + blockSize - 1) / blockSize);
            int *dev_idata;

            hipMalloc((void **)&dev_idata, size * sizeof(int));

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            // TODO
            // UpSweep
            for (int d = 0; d < ilog2ceil(size); d++) {
                kernUpSweep<<<blockPerGrids, blockSize>>>(n, d, dev_idata);
                hipDeviceSynchronize(); // ensure that the previous cuda jobs have completed
            }
            // set the last value of dev_idata to zero
            hipMemset(dev_idata + size - 1, 0, sizeof(int));

            // DownSweep
            for (int d = ilog2ceil(size) - 1; d >= 0; d--) {
                kernDownSweep<<<blockPerGrids, blockSize>>>(n, d, dev_idata);
                hipDeviceSynchronize();
            }
            timer().endGpuTimer();
            hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_idata);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int *dev_bools;
            int *dev_indices;
            int *dev_idata;
            int *dev_odata;
            int size = twoPowHost(ilog2ceil(n));
            int cnt = 0;

            dim3 blockPerGrids((n + blockSize - 1) / blockSize);
            dim3 fullBlockPerGrids((size + blockSize - 1) / blockSize);

            hipMalloc((void **)&dev_bools, size * sizeof(int));
            hipMalloc((void **)&dev_indices, size * sizeof(int));
            hipMalloc((void **)&dev_idata, size * sizeof(int));
            hipMalloc((void **)&dev_odata, size * sizeof(int));

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            // TODO
            Common::kernMapToBoolean<<<blockPerGrids, blockSize>>>(n, dev_bools, dev_idata);
            hipDeviceSynchronize();
            hipMemcpy(dev_indices, dev_bools, n * sizeof(int), hipMemcpyDeviceToDevice);

            // scan
            for (int d = 0; d < ilog2ceil(size); d++) {
                kernUpSweep<<<fullBlockPerGrids, blockSize>>>(n, d, dev_indices);
                hipDeviceSynchronize();
            }

            hipMemset(dev_indices + size - 1, 0, sizeof(int));

            for (int d = ilog2ceil(size) - 1; d >= 0; d--) {
                kernDownSweep<<<fullBlockPerGrids, blockSize>>>(n, d, dev_indices);
                hipDeviceSynchronize();
            }

            Common::kernScatter<<<blockPerGrids, blockSize>>>(n, dev_odata, dev_idata, dev_bools, dev_indices);
            timer().endGpuTimer();

            hipMemcpy(&cnt, dev_indices + size - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(odata, dev_odata, cnt * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_indices);
            hipFree(dev_bools);

            return cnt;
        }
    }
}
