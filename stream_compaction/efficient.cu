#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <iostream>
#include <>


namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */

        __global__ void kernUpsweepStep(int n, int destStride, int srcStride, int *data) {
            int idx = threadIdx.x + (blockIdx.x * blockDim.x);
            int actualIdx = (idx + 1) * destStride - 1;
            if (actualIdx >= n) {
                return;
            }
            data[actualIdx] += data[actualIdx - srcStride];
        }

        __global__ void kernDownsweepStep(int n, int destStride, int srcStride, int* data) {
            int idx = threadIdx.x + (blockIdx.x * blockDim.x);
            int actualIdx = (idx + 1) * destStride - 1;
            if (actualIdx >= n) {
                return;
            }
            int temp = data[actualIdx - srcStride];
            data[actualIdx - srcStride] = data[actualIdx];
            data[actualIdx] += temp;
        }

        void scanWithoutTimer(int n, dim3 blocksPerGrid, int* dev_data) {
            // TODO

            for (int d = 0; d <= ilog2ceil(n); d++) {
                kernUpsweepStep << <blocksPerGrid, blockSize >> > (n, std::pow(2, d + 1), std::pow(2, d), dev_data);
                hipDeviceSynchronize();
            }

            int zero = 0;
            hipMemcpy(dev_data + n - 1, &zero, sizeof(int), hipMemcpyHostToDevice);

            for (int d = ilog2ceil(n); d >= 0; d--) {
                kernDownsweepStep << <blocksPerGrid, blockSize >> > (n, std::pow(2, d + 1), std::pow(2, d), dev_data);
                hipDeviceSynchronize();
            }
        }

        int closestPower(int num) {
            int i = 0;
            while (num > std::pow(2, i)) {
                i++;
            }
            return std::pow(2, i);
        }

        int* zeros(int num) {
            int *arr = (int*)malloc(num * sizeof(int));
            for (int i = 0; i < num; i++) {
                arr[i] = 0;
            }
            return arr;
        }

        void scan(int n, int *odata, const int *idata) {
            int nPot = closestPower(n);

            dim3 fullBlocksPerGrid((nPot + blockSize - 1) / blockSize);

            int* dev_data;

            hipMalloc((void**)&dev_data, nPot * sizeof(int));
            checkCUDAError("Error during hipMalloc dev_data");

            hipMemcpy(dev_data + nPot - n, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("Error during hipMemcpy idata ==> dev_data");

            int* zero = zeros(n);

            hipMemcpy(dev_data, zero, (nPot - n) * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("Error during hipMemcpy zero ==> dev_data");

            hipDeviceSynchronize();

            timer().startGpuTimer();

            scanWithoutTimer(nPot, fullBlocksPerGrid, dev_data);
            
            timer().endGpuTimer();

            hipMemcpy(odata, dev_data + nPot - n, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("Error during hipMemcpy odata");

            hipFree(dev_data);
            checkCUDAError("Error during hipFree dev_data");

            free(zero);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int nPot = closestPower(n);
            
            dim3 fullBlocksPerGrid((nPot + blockSize - 1) / blockSize);

            int* dev_idata, * dev_bools, * dev_indices, int* dev_odata;

            hipMalloc((void**)&dev_idata, nPot * sizeof(int));
            checkCUDAError("Error during hipMalloc dev_idata");

            hipMalloc((void**)&dev_bools, nPot * sizeof(int));
            checkCUDAError("Error during hipMalloc dev_bools");

            hipMalloc((void**)&dev_indices, nPot * sizeof(int));
            checkCUDAError("Error during hipMalloc dev_indices");

            hipMalloc((void**)&dev_odata, nPot * sizeof(int));
            checkCUDAError("Error during hipMalloc dev_odata");

            hipMemcpy(dev_idata + nPot - n, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("Error during hipMemcpy dev_data");

            int* zero = zeros(n);

            hipMemcpy(dev_idata, zero, (nPot - n) * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("Error during hipMemcpy zero ==> dev_data");

            hipDeviceSynchronize();
            
            timer().startGpuTimer();
            //// TODO
            //

            StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (nPot, dev_bools, dev_idata);

            hipMemcpy(dev_indices, dev_bools, nPot * sizeof(int), hipMemcpyDeviceToDevice);
            checkCUDAError("Error during hipMemcpy dev_data");

            scanWithoutTimer(nPot, fullBlocksPerGrid, dev_indices);
            StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> > (n, dev_odata + nPot - n, dev_idata + nPot - n, dev_bools + nPot - n, dev_indices + nPot - n);

            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata + nPot - n, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("Error during hipMemcpy dev_odata");

            int count = 0;
            int lastbool = 0;
            hipMemcpy(&count, dev_indices + nPot - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&lastbool, dev_bools + nPot - 1, sizeof(int), hipMemcpyDeviceToHost);

            count += lastbool;

            hipFree(dev_odata);
            checkCUDAError("Error during hipFree dev_odata");

            hipFree(dev_indices);
            checkCUDAError("Error during hipFree dev_indices");

            hipFree(dev_bools);
            checkCUDAError("Error during hipFree dev_bools");

            hipFree(dev_idata);
            checkCUDAError("Error during hipFree dev_idata");

            return count;
        }
    }
}
