#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 128 

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        __global__ void kernUpsweep(int n, int depth, int* odata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            int path = pow(2, depth);
            if (index >= n || index % (2*path) != 0 ||(index + 2*path-1) >= n) {
                return;
            }
            odata[index + 2*path - 1] += odata[index + path - 1];
            return;
        }

        __global__ void kernDownsweep(int n, int depth, int maxDepth, int* odata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            int path = pow(2, depth);
            if (index == n - 1 && depth == maxDepth) {
                odata[index] = 0;//have to do this here because cant do this in void directly
                return;
            }
            if (index >= n || index % (2 * path) != 0 || (index+2*path-1) >= n) {
                return;
            }
            int saveVal = odata[index + path - 1];
            odata[index + path - 1] = odata[index + 2 * path - 1];
            odata[index + 2 * path - 1] += saveVal;
            return;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            //fill array and initialization
            int maxN = pow(2, ilog2ceil(n));
            dim3 blockDim((maxN + blockSize - 1) / blockSize);
            int* odataMax;
            hipMalloc((void**)&odataMax, maxN * sizeof(int));
            hipMemcpy(odataMax, idata, n * sizeof(int), hipMemcpyHostToDevice);
            hipMemset(odataMax + n, 0, (maxN - n) * sizeof(int));

            timer().startGpuTimer();
            // TODO
            //Upsweep
            for (int i = 0; i < ilog2ceil(n); i++) {
                kernUpsweep <<<blockDim, blockSize >> > (n, i, odataMax);
            }
            //Downsweep
            for (int i = ilog2ceil(n) - 1; i > 0; i--) {
                kernDownsweep << <blockDim, blockSize >> > (n, i, maxN, odataMax);
            }
            timer().endGpuTimer();

            hipMemcpy(odata, odataMax, n*sizeof(int), hipMemcpyDeviceToHost);
            hipFree(odataMax);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            //initialization
            int maxN = pow(2, ilog2ceil(n));
            dim3 blockDim((n + blockSize - 1) / blockSize);
            int* odataMax, *oBoll, * oScan;
            hipMalloc((void**)&odataMax, maxN * sizeof(int));
            hipMemcpy(odataMax, idata, n * sizeof(int), hipMemcpyHostToDevice);
            hipMemset(odataMax + n, 0, (maxN - n) * sizeof(int));
            hipMalloc((void**)&oBoll, maxN * sizeof(int));
            hipMalloc((void**)&oScan, maxN * sizeof(int));

            timer().startGpuTimer();
            // TODO
            //first create temp array
            StreamCompaction::Common::kernMapToBoolean << <blockDim, blockSize >> > (n, oBoll, idata);
            hipMemcpy(oScan, oBoll, maxN * sizeof(int), hipMemcpyDeviceToDevice);
            //Upsweep
            for (int i = 0; i < ilog2ceil(n); i++) {
                kernUpsweep << <blockDim, blockSize >> > (n, i, oScan);
            }
            //Downsweep
            for (int i = ilog2ceil(n) - 1; i > 0; i--) {
                kernDownsweep << <blockDim, blockSize >> > (n, i, maxN, oScan);
            }
            //scatter
            StreamCompaction::Common::kernScatter << <blockDim, blockSize >> > (n, odata, idata, odataMax, oScan);
            timer().endGpuTimer();

            //now we get the last index of oScan to return;
            int* lastIndex = new int[1];
            hipMemcpy(lastIndex, oScan + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipFree(oBoll);
            hipFree(oScan);
            hipFree(odataMax);
            return lastIndex[0];//how do we get the final index?
        }
    }
}
