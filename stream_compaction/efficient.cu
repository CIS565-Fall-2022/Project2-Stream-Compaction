#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 128

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // unlike naive impl, this one doesn't shift the array
        __global__ void kernPadArray(int n, int paddedLen, int* odata, int* idata) {
          int index = threadIdx.x + (blockIdx.x * blockDim.x);
          if (index < n) {
            odata[index] = idata[index];
          }
          else if (index < paddedLen) {
            odata[index] = 0;
          }
        }

        int* dev_unpadded_idata;
        int* dev_idata;
        int* dev_odata;

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            // Pad array
            int exponent = ilog2ceil(n);
            int paddedLength = pow(2, exponent);
            dim3 fullBlocksPerGrid((paddedLength + blockSize - 1) / blockSize);

            hipMalloc((void**)&dev_unpadded_idata, n * sizeof(int));
            hipMalloc((void**)&dev_idata, paddedLength * sizeof(int));
            hipMalloc((void**)&dev_odata, paddedLength * sizeof(int));

            hipMemcpy(dev_unpadded_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorFn("Cuda memcpy idata no work");

            kernPadArray << <fullBlocksPerGrid, blockSize >> > (n, paddedLength, dev_idata, dev_unpadded_idata);
            printCudaArray(paddedLength, dev_idata);

            // Build tree

            hipFree(dev_unpadded_idata);
            hipFree(dev_idata);
            hipFree(dev_odata);

            timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
