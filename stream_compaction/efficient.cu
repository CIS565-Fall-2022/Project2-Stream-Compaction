#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int t, int* data) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index < n) {
                data[t * index + t - 1] += data[t * index + (t >> 1) - 1];
            }
        }
        __global__ void kernDownSweep(int n, int t, int* data) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index < n) {
                int tmp = data[t * index + (t >> 1) - 1];
                data[t * index + (t >> 1) - 1] = data[t * index + t - 1];
                data[t * index + t - 1] += tmp;
            }
        }
        __global__ void kernSetArray(int value, int index, int* data) {
            data[index] = value;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int blockSize = 128;
            int* buffer;
            int N = 1 << ilog2ceil(n);
            hipMalloc((void**)&buffer, N * sizeof(int));
            hipMemcpy(buffer, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            for (int d = 0; d < ilog2ceil(n); d++) {
                int computeCount = n >> d + 1;  //up sweep is n, down sweep is N. It's not a typo
                dim3 blockCount((computeCount + blockSize - 1) / blockSize);
                kernUpSweep << <blockCount, blockSize >> > (computeCount, 1<<d+1, buffer);  //todo non power of 2
            }
            //hipMemset(buffer + N - 1, 0, sizeof(int)); it cost to much
            kernSetArray << <1, 1 >> > (0, N - 1, buffer);
            for (int d = ilog2ceil(n) - 1; d >= 0; d--) {
                int computeCount = N >> d + 1;
                dim3 blockCount((computeCount + blockSize - 1) / blockSize);
                kernDownSweep << <blockCount, blockSize >> > (computeCount, 1 << d+1, buffer);
            }
            timer().endGpuTimer();

            hipMemcpy(odata, buffer, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(buffer);
        }


        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int* bools, * indices, * ibuffer, * obuffer;
            hipMalloc((void**)&bools, n * sizeof(int));
            hipMalloc((void**)&indices, n * sizeof(int));
            hipMalloc((void**)&ibuffer, n * sizeof(int));
            hipMalloc((void**)&obuffer, n * sizeof(int));
            hipMemcpy(ibuffer, idata, n * sizeof(int), hipMemcpyHostToDevice);

            int blockSize = 128;
            dim3 blockCount((n + blockSize - 1) / blockSize);

            timer().startGpuTimer();
            Common::kernMapToBoolean<<<blockCount, blockSize>>>(n, bools, ibuffer);

            //------------------ scan ---------------------
            int N = 1 << ilog2ceil(n);
            hipMemcpy(indices, bools, n * sizeof(int), hipMemcpyDeviceToDevice);

            for (int d = 0; d < ilog2ceil(n); d++) {
                int computeCount = n >> d + 1;  //up sweep is n, down sweep is N. It's not a typo
                dim3 blockCount((computeCount + blockSize - 1) / blockSize);
                kernUpSweep << <blockCount, blockSize >> > (computeCount, 1 << d + 1, indices);  //todo non power of 2
            }
            kernSetArray << <1, 1 >> > (0, N - 1, indices);
            for (int d = ilog2ceil(n) - 1; d >= 0; d--) {
                int computeCount = N >> d + 1;
                dim3 blockCount((computeCount + blockSize - 1) / blockSize);
                kernDownSweep << <blockCount, blockSize >> > (computeCount, 1 << d + 1, indices);
            }
            //------------------ scan ---------------------

            Common::kernScatter << <blockCount, blockSize >> > (n, obuffer, ibuffer, bools, indices);
            timer().endGpuTimer();

            int count, bias;
            hipMemcpy(&count, indices + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&bias, bools + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            int remainNum = count + bias;
            hipMemcpy(odata, obuffer, remainNum * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(bools);
            hipFree(indices);
            hipFree(ibuffer);
            hipFree(obuffer);
            return remainNum;   //todo
        }
    }
}
