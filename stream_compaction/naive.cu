#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "common.h"
#include "naive.h"
#include <>
#include <hip/device_functions.h>
#include <iostream>

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
        
        __global__ void kernScanStep(int n, int stride, int* idata, int* odata) {
            int idx = threadIdx.x + (blockIdx.x * blockDim.x);
            if (idx >= n) {
                return;
            }
            odata[idx] = idata[idx];
            if (idx < stride) {
                return;
            }
            odata[idx] += idata[idx - stride];
        }
        
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            int* dev_idata, * dev_odata;

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("Error during hipMalloc dev_idata");

            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("Error during hipMalloc dev_odata");

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("Error during hipMemcpy dev_idata");
            
            hipDeviceSynchronize();

            // TODO

            timer().startGpuTimer();

            for (int d = 0; d <= ilog2ceil(n); d++) { 
                kernScanStep << <fullBlocksPerGrid, blockSize >> > (n, std::pow(2, d), dev_idata, dev_odata);
                hipDeviceSynchronize();

                hipMemcpy(dev_idata, dev_odata, n * sizeof(int), hipMemcpyDeviceToDevice);
                checkCUDAError("Error during hipMemcpy dev_odata ==> dev_idata");
            }

            timer().endGpuTimer();

            hipMemcpy(odata + 1, dev_odata, (n - 1) * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("Error during hipMemcpy odata");
            
            hipFree(dev_idata);
            checkCUDAError("Error during hipFree dev_idata");
            hipFree(dev_odata);
            checkCUDAError("Error during hipFree dev_odata");


            hipDeviceSynchronize();
        }

    }
}
