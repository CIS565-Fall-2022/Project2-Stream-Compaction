#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#define blockSize 128

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
        __global__ void KernNaiveScanIteration(int n, int d, int* odata, int* idata)
        {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n )
            {
                return;
            }
            if (index >= pow(2, d - 1))
                odata[index] = idata[index - (int)pow(2, d - 1)] + idata[index];
            else
                odata[index] = idata[index];
        }


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // TODO
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            int* dev_odata1;
            int* dev_odata2;
            hipMalloc((void**)&dev_odata1, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata1 failed!");
            hipMalloc((void**)&dev_odata2, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata1 failed!");

            hipMemcpy(dev_odata1, idata, sizeof(int) * n, hipMemcpyHostToDevice);

            timer().startGpuTimer();
            for (int d = 1; d <= ilog2ceil(n); ++d)
            {
                if (d % 2 == 1)
                {
                    KernNaiveScanIteration << <fullBlocksPerGrid, blockSize >> > (n, d, dev_odata2, dev_odata1);
                }
                else
                {
                    KernNaiveScanIteration << <fullBlocksPerGrid, blockSize >> > (n, d, dev_odata1, dev_odata2);
                }
            }

            odata[0] = 0;
            hipMemcpy(odata + 1, dev_odata1, sizeof(int) * (n - 1), hipMemcpyDeviceToHost);
            timer().endGpuTimer();


            //Clean up
            hipFree(dev_odata1);
            hipFree(dev_odata2);

        }
    }
}
