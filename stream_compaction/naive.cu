#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#define blockSize 128

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
        __global__ void kernNaiveScan(int n, int* odata, int* idata, int stride) 
        {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            if (index >= stride) {
                odata[index] = idata[index - stride] + idata[index];
            }
            else {
                odata[index] = idata[index];
            }
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_idata;
            int* dev_odata;

            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            // TODO
            for (int d = 1; d <= ilog2ceil(n); d++) {
                kernNaiveScan << < fullBlocksPerGrid, blockSize >> > (n, dev_odata, dev_idata, pow(2.0,d-1));
                
                // ping-pong buffer
                int* tmp = dev_idata;
                dev_idata = dev_odata;
                dev_odata = tmp;
            }

            timer().endGpuTimer();

            // covert from inclusive scan to exclusive scan
            // copy the memory from the second index and manually set identity to the first element
            hipMemcpy(odata + 1, dev_idata, (n-1) * sizeof(int), hipMemcpyDeviceToHost);
            odata[0] = 0;

            // free cuda memory
            hipFree(dev_odata);
            hipFree(dev_idata);
        }
    }
}
