#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive { 
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__

        __global__ void kernPartialScan(int* out, int* in, int n, int stride) {
            int idx = blockDim.x * blockIdx.x + threadIdx.x;
            if (idx >= n) {
                return;
            }
            out[idx] = in[idx] + in[idx - stride];
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            size_t bytes = n * sizeof(int);
            int* buf, * tmp;
            hipMalloc(&buf, bytes);
            hipMalloc(&tmp, bytes);
            hipMemcpy(buf, idata, bytes, hipMemcpyKind::hipMemcpyHostToDevice);

            int stride = 1;
            while (stride < n) {
                int num = n - stride;
                int blockSize = Common::getDynamicBlockSizeEXT(num);
                int blockNum = (num + blockSize - 1) / blockSize;
                kernPartialScan<<<blockNum, blockSize>>>(tmp + stride, buf + stride, n - stride, stride);
                hipMemcpy(buf + stride, tmp + stride, num * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToDevice);
                stride <<= 1;
            }
            odata[0] = 0;
            hipMemcpy(odata + 1, buf, (n - 1) * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
            hipFree(buf);
            hipFree(tmp);

            timer().endGpuTimer();
        }
    }
}
