#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;

        #define blockSize 8

        int* dev_idata;
        int* dev_odata;
        int* dev_buf;

        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // TODO: __global__

        __global__ void kernScan(int N, int* idata, int* odata, int depth) {
            int k = threadIdx.x + (blockIdx.x * blockDim.x);
            if (k >= N) {
                return;
            }

            if (k >= 1 << (depth - 1)) {
                odata[k] = idata[k - (1 << (depth - 1))] + idata[k];
            }
            else {
                odata[k] = idata[k];
            }
            
        }

        void zeroArray(int n, int* a) {
            for (int i = 0; i < n; i++) {
                a[i] = 0;
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

            int arrLen;
            int maxDepth = ilog2ceil(n);
            maxDepth > ilog2(n) ? arrLen = pow(2, maxDepth) : arrLen = n;
            zeroArray(arrLen, odata);

            dim3 threadsPerBlock(arrLen/blockSize);

            int* buf = new int[arrLen];

            for (int i = 0; i < arrLen; i++) {
                if (i < n) {
                    buf[i] = idata[i];
                }
                else {
                    buf[i] = 0;
                }
            }

            // TODO
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            hipMalloc((void**)&dev_odata, arrLen * sizeof(int));
            hipMalloc((void**)&dev_buf, arrLen * sizeof(int));

            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            hipMemcpy(dev_odata, odata, sizeof(int) * arrLen, hipMemcpyHostToDevice);
            hipMemcpy(dev_buf, buf, sizeof(int) * arrLen, hipMemcpyHostToDevice);

            for (int i = 1; i <= maxDepth; i++) {
                kernScan << <threadsPerBlock, blockSize >> > (arrLen, dev_buf, dev_odata, i);
                hipDeviceSynchronize();
                hipMemcpy(dev_buf, dev_odata, sizeof(int) * arrLen, hipMemcpyDeviceToDevice);
            }         

            hipMemcpy((void**)idata, dev_idata, sizeof(int) * n, hipMemcpyDeviceToHost);
            hipMemcpy(odata, dev_odata, sizeof(int) * arrLen, hipMemcpyDeviceToHost);
            hipMemcpy(buf, dev_buf, sizeof(int) * arrLen, hipMemcpyDeviceToHost);

            hipFree(dev_idata);
            hipFree(dev_odata); 
            hipFree(dev_buf);

            timer().endGpuTimer();
        }
    }
}
