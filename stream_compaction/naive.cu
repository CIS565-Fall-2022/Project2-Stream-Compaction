#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#include <>

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        constexpr unsigned blockSize = 128; // TODO test different blockSizes

        __global__ void kernPrefixSumExclusiveScan(int d, int n, int *idata, int *odata) {
            unsigned index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n) { return; }

            int odataidx = index; // clusmy but forces exclusive scan behavior
            if (d == 1) {
                if (++odataidx >= n) { return; }
            }

            unsigned cutoff = 1 << d - 1;
            odata[odataidx] = idata[index];
            if (index >= cutoff) {
                odata[odataidx] += idata[index - cutoff];
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_data1;
            int* dev_data2;
            hipMalloc((void**)&dev_data1, n * sizeof(int));
            hipMalloc((void**)&dev_data2, n * sizeof(int));
            hipMemcpy(dev_data1, idata, n * sizeof(int), hipMemcpyHostToDevice);
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            timer().startGpuTimer();
            int d;
            for (d = 1; d <= ilog2ceil(n); ++d) {
                kernPrefixSumExclusiveScan<<<fullBlocksPerGrid, blockSize>>>(d, n, dev_data1, dev_data2);
                std::swap(dev_data1, dev_data2); // swap i/o arrays for next summing
                hipDeviceSynchronize();
            }
            timer().endGpuTimer();

            // ensure we send back the last output bufer
            d % 2 == 0 ? 
                hipMemcpy(odata, dev_data1, n * sizeof(int), hipMemcpyDeviceToHost) :
                hipMemcpy(odata, dev_data2, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_data1);
            hipFree(dev_data2);
        }
    }
}
