#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#define blockSize 128 

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
        __global__ void kernScan(int n, int depth, int* odata, const int* idata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }
            int index1 = 1 << depth;
            //int index1 = 1 << depth;
            odata[index] = idata[index];
            if (index >= index1) {
                odata[index] += idata[index - index1];
            }
            return;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            dim3 blockDim((n + blockSize - 1) / blockSize);
            int depth = ilog2ceil(n);
            int* input, *output;
            hipMalloc((void**)&input, n*sizeof(int));
            hipMalloc((void**)&output, n*sizeof(int));
            hipMemcpy(input, idata, n*sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(output, idata, n * sizeof(int), hipMemcpyHostToDevice);


            timer().startGpuTimer();
            for (int i = 0; i <= depth; ++i) {
                kernScan<<<blockDim, blockSize>>>(n, i, output, input);
                checkCUDAError("kernScan failed");
                //std::swap(input, output);
                hipMemcpy(input, output, n * sizeof(int), hipMemcpyDeviceToDevice);
            }

            timer().endGpuTimer();

            //hipMemcpy(odata, output, n, hipMemcpyDeviceToHost);
            //change from inclusive to excluvise
            hipMemcpy(odata + 1, output, (n - 1)*sizeof(int), hipMemcpyDeviceToHost);
            odata[0] = 0;
            hipFree(input);
            hipFree(output);
        }
    }
}
