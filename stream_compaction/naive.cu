#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#include <iostream> // PLEASE REMOVE THIS AFTER TESTING

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        /**
         * Shifts the array one to the right.
         */
        __global__ void kernInclToExcl(int n, int* odata, const int* idata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            odata[index] = (index > 0) ? idata[index - 1] : 0;
        }
        
        /**
         * Performs one thread's work for a naive parallel prefix-sum.
         */
        __global__ void kernNaiveScan(int n, int d, int offset, int* odata, const int* idata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            if (index >= offset) {
                odata[index] = idata[index - offset] + idata[index];
            }
            else {
                odata[index] = idata[index];
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // Determine block size
            dim3 blocksPerGrid((n + blockSize - 1) / blockSize);

            // Allocate seperate arrays to hold results between iterations
            int* dev_scan_input;
            int* dev_scan_output;

            // Allocate device memory
            hipMalloc((void**)&dev_scan_input, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_scan_input failed!");
            hipMalloc((void**)&dev_scan_output, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_scan_output failed!");

            // Copy data to the GPU
            hipMemcpy(dev_scan_input, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            hipMemcpy(dev_scan_output, odata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAErrorFn("memcpy to GPU failed!");

            timer().startGpuTimer();
            // Transform inclusive array to exclusive array
            kernInclToExcl << <blocksPerGrid, blockSize >> > (n, dev_scan_output, dev_scan_input);

            // Perform exclusive scan
            int log2_n = ilog2ceil(n);
            int offset = 1;
            for (int d = 1; d <= log2_n; ++d) {
                kernNaiveScan << <blocksPerGrid, blockSize >> > (n, d, offset, dev_scan_input, dev_scan_output);
                std::swap(dev_scan_input, dev_scan_output);
                offset *= 2;
            }
            timer().endGpuTimer();

            // Copy data back to the CPU
            hipMemcpy(odata, dev_scan_output, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAErrorFn("memcpy to CPU failed!");

            //// Print output vals
            //std::cout << "After scan: " << std::endl;
            //for (int i = 0; i < n; i++) {
            //    std::cout << "  scan[" << i << "]: " << odata[i] << std::endl;
            //}

            // Cleanup memory
            hipFree(dev_scan_input);
            hipFree(dev_scan_output);
            checkCUDAErrorFn("hipFree failed!");
        }
    }
}
