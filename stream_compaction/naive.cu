#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

const int blockSize = 128;

__device__ inline int twoPow(int d) {
    return (1 << (d));
}

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__

        __global__ void kernNaiveScan(int n, int d, int *odata, int *idata) {
            int idx = blockDim.x * blockIdx.x + threadIdx.x;
            if (idx >= n) return;
            // Add adjacent elements to get the prefix sum
            if (idx >= twoPow(d))
                odata[idx] = idata[idx] + idata[idx - twoPow(d)];
            else
                odata[idx] = idata[idx];
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int *dev_idata;
            int *dev_odata;
            dim3 blocksPerGrid((n + blockSize - 1) / blockSize);
            // allocate
            hipMalloc((void **)&dev_idata, n * sizeof(int));
            checkCUDAError("allcoate dev_idata failed!\n");
            hipMalloc((void **)&dev_odata, n * sizeof(int));
            checkCUDAError("allcoate dev_odata failed!\n");

            // move data to device
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            
            timer().startGpuTimer();
            // TODO: Naive Scan
            for (int d = 0; d < ilog2ceil(n); d++) {
                kernNaiveScan<<<blocksPerGrid, blockSize>>>(n, d, dev_odata, dev_idata);
                std::swap(dev_odata, dev_idata);
            }
            timer().endGpuTimer();

            // shift right
            odata[0] = 0;
            hipMemcpy(odata + 1, dev_idata, (n - 1) * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_idata);
            hipFree(dev_odata);
        }
    }
}
