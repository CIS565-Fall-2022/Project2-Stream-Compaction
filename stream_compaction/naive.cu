#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#define blockSize 128 

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
        __global__ void kernScan(int n, int depth, int* odata, const int* idata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }
            odata[index] = idata[index];
            if (index >= int(pow(2, depth))) {
                odata[index] += idata[index - int(pow(2, depth ))];
            }
            return;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {

            dim3 blockDim((n + blockSize - 1) / blockSize);
            int depth = ilog2ceil(n);
            bool oddEvenCount = false;
            int* input, * output;
            hipMalloc((void**)&input, n*sizeof(int));
            hipMalloc((void**)&output, n*sizeof(int));
            hipMemcpy(input, idata, n, hipMemcpyHostToDevice);


            timer().startGpuTimer();
            for (int i = 0; i < depth; i++) {
                
                kernScan<<<blockDim, blockSize>>>(n, i, output, input);
                std::swap(output, input);
                oddEvenCount = !oddEvenCount;
            }
            

            timer().endGpuTimer();

            if (!oddEvenCount) {
                std::swap(input, output);
            }
            //hipMemcpy(odata, output, n, hipMemcpyDeviceToHost);
            //change from inclusive to excluvise
            hipMemcpy(odata + 1, output, n - 1, hipMemcpyDeviceToHost);
            odata[0] = 0;
            hipFree(input);
            hipFree(output);
        }
    }
}
